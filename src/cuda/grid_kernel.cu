#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "cuda/grid_kernel.h"
#include <cstdio>
#include <string>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

#include <cfloat>

#define CUDA_CHECK_ERROR(); {hipError_t error=hipGetLastError(); if(error){printf("CUDA error: %d (%s) in %s line %d\n",error,hipGetErrorString(error),__FILE__,__LINE__);exit(-1);}}


__global__ void TransformPoints(double *outx, double *outy, double *outz, double m[16], double *x, double *y, double *z,int size)
{
	int i=threadIdx.x+blockIdx.x*1024+blockIdx.y*1024*1024;


	if(i<size)
	{
		double xx=x[i];
		double yy=y[i];
		double zz=z[i];

		outx[i]=m[0]*xx+m[4]*yy+m[8]*zz+m[12];
		outy[i]=m[1]*xx+m[5]*yy+m[9]*zz+m[13];
		outz[i]=m[2]*xx+m[6]*yy+m[10]*zz+m[14];
	}
}

__global__ void TransformPoints(double *outx, double *outy, double *outz, double m[16], int size)
{
	int i=threadIdx.x+blockIdx.x*1024+blockIdx.y*1024*1024;


	if(i<size)
	{
		double xx=outx[i];
		double yy=outy[i];
		double zz=outz[i];

		outx[i]=m[0]*xx+m[4]*yy+m[8]*zz+m[12];
		outy[i]=m[1]*xx+m[5]*yy+m[9]*zz+m[13];
		outz[i]=m[2]*xx+m[6]*yy+m[10]*zz+m[14];
	}
}
__global__ void TransformPoints(double *outxyz, double m[16], double *xyz,int size)
{
	int i=threadIdx.x+blockIdx.x*1024+blockIdx.y*1024*1024;


	if(i<size)
	{
		double xx=xyz[3*i+0];
		double yy=xyz[3*i+1];
		double zz=xyz[3*i+2];

		outxyz[3*i+0]=m[0]*xx+m[4]*yy+m[8]*zz+m[12];
		outxyz[3*i+1]=m[1]*xx+m[5]*yy+m[9]*zz+m[13];
		outxyz[3*i+2]=m[2]*xx+m[6]*yy+m[10]*zz+m[14];
	}
}

__global__ void TransformPoints(double *outxyz, double m[16], int size)
{
	int i=threadIdx.x+blockIdx.x*1024+blockIdx.y*1024*1024;


	if(i<size)
	{
		double xx=outxyz[3*i+0];
		double yy=outxyz[3*i+1];
		double zz=outxyz[3*i+2];

		outxyz[3*i+0]=m[0]*xx+m[4]*yy+m[8]*zz+m[12];
		outxyz[3*i+1]=m[1]*xx+m[5]*yy+m[9]*zz+m[13];
		outxyz[3*i+2]=m[2]*xx+m[6]*yy+m[10]*zz+m[14];
	}
}


void cudaTransformScan(double *destx, double *desty, double *destz, double *mat, double *srcx, double *srcy, double *srcz, unsigned int size)
{
	int by=1+int(size/(1024*1024.0));
	dim3 blocks(1024,by),threads(1024);
	//printf("TransformPoints<<<(%d,%d,%d),(%d,%d,%d)>>>\n",blocks.x,blocks.y,blocks.z,threads.x,threads.y,threads.z);
	
	hipDeviceSynchronize();
	CUDA_CHECK_ERROR();
	TransformPoints<<<blocks,threads>>>(destx,desty,destz,mat,srcx,srcy,srcz,size);
	CUDA_CHECK_ERROR();
	hipDeviceSynchronize();
	CUDA_CHECK_ERROR();
}

void cudaTransformScan(double *destx, double *desty, double *destz, double *mat, unsigned int size)
{
	int by=1+int(size/(1024*1024.0));
	dim3 blocks(1024,by),threads(1024);
	
	hipDeviceSynchronize();
	CUDA_CHECK_ERROR();
	TransformPoints<<<blocks,threads>>>(destx,desty,destz,mat,size);
	CUDA_CHECK_ERROR();
	hipDeviceSynchronize();
	CUDA_CHECK_ERROR();
}
void cudaTransformScan(double *destxyz, double *mat, double *srcxyz, unsigned int size)
{
	int by=1+int(size/(1024*1024.0));
	dim3 blocks(1024,by),threads(1024);
	
	hipDeviceSynchronize();
	CUDA_CHECK_ERROR();
	TransformPoints<<<blocks,threads>>>(destxyz,mat,srcxyz,size);
	CUDA_CHECK_ERROR();
	hipDeviceSynchronize();
	CUDA_CHECK_ERROR();
}
void cudaTransformScan(double *destxyz, double *mat, unsigned int size)
{
	int by=1+int(size/(1024*1024.0));
	dim3 blocks(1024,by),threads(1024);
	
	hipDeviceSynchronize();
	CUDA_CHECK_ERROR();
	TransformPoints<<<blocks,threads>>>(destxyz,mat,size);
	CUDA_CHECK_ERROR();
	hipDeviceSynchronize();
	CUDA_CHECK_ERROR();
}

void getKernelDim(dim3 &_block, dim3 &_thread, int num_points)
{
	_block.x = 1024;	_block.y = 1; _block.z = ceil(num_points/double(1024*1024.0));
	_thread.x = 1024; _thread.y = 1; _thread.z = 1;

}
/*
void getKernelDimBuckets(dim3 &_block, dim3 &_thread, int num_buckets)
{
	_block.x = num_buckets;  _block.y = num_buckets; _block.z = 1;
	_thread.x = num_buckets; _thread.y = 1; _thread.z = 1;
}

*/

__device__ int getThreadId(int bx, int by, int bz, int tx, int ty, int tz)
{
	return tx + bx*1024 + bz*1024*1024;
}

__global__ void
kernel_ComputeIndexes(unsigned int *_d_index, double *_x, double *_y, double *_z, 
					 int _amountOfPoints, int num_buckets)
{
	int index = getThreadId(blockIdx.x,blockIdx.y,blockIdx.z, threadIdx.x, threadIdx.y,threadIdx.z);

	if(index < _amountOfPoints)
	{
		double x = _x[index];
		double y = _y[index];
		double z = _z[index];

		double border;
		int curr_ind;
		double cur_two_power_x;

		int upto, diff;

		if(num_buckets == 1024){upto = 10; diff = 1073741823;}
		if(num_buckets == 512){upto = 9; diff   = 134217727;}
		if(num_buckets == 256){upto = 8; diff   = 16777215;}
		if(num_buckets == 128){upto = 7; diff   = 2097151;}
		if(num_buckets == 64){upto = 6; diff    = 262143;}
		if(num_buckets == 32){upto = 5; diff    = 32767;}
		if(num_buckets == 16){upto = 4; diff    = 4095;}
		///////////////
		border = 0;
		curr_ind = 0;
		cur_two_power_x = 0;
		//__syncthreads();
	
		cur_two_power_x = 1.0f;
		for(int kk=0;kk<upto;kk++)
		{
		cur_two_power_x = cur_two_power_x / 2.0f;
			if(x <= border)
			{	
			curr_ind = 2 * curr_ind + 1; 
			border = border - cur_two_power_x;
			}else
			{
			curr_ind = 2 * curr_ind + 2;
			border = border + cur_two_power_x;	
			};
		}
			
		border = 0.0f;
		cur_two_power_x = 1.0f;
		for(int kk=0;kk<upto;kk++)
		{
			cur_two_power_x = cur_two_power_x / 2.0f;	
			if(y <= border)
			{	
			curr_ind = 2 * curr_ind + 1; 
			border = border - cur_two_power_x;
			}else
			{
			curr_ind = 2 * curr_ind + 2;
			border = border + cur_two_power_x;	
			};
		}
	
	
		border = 0.0f;
		cur_two_power_x = 1.0f;
		for(int kk=0;kk<upto;kk++)
		{
			cur_two_power_x = cur_two_power_x / 2.0f;	
			if(z <= border)
			{	
			curr_ind = 2 * curr_ind + 1; 
			border = border - cur_two_power_x;
			}else
			{
			curr_ind = 2 * curr_ind + 2;
			border = border + cur_two_power_x;	
			};
		}
	
		curr_ind = curr_ind-diff;//16777215;//SUM_INDEXES_IN_LEVELS;
		_d_index[index] = curr_ind;
		///////////////
	}
}

__global__ void
kernel_ComputeIndexes(unsigned int *_d_index, double *xyz, 
					 int _amountOfPoints, int num_buckets)
{
	int index = getThreadId(blockIdx.x,blockIdx.y,blockIdx.z, threadIdx.x, threadIdx.y,threadIdx.z);

	if(index < _amountOfPoints)
	{
		double x = xyz[3*index+0];
		double y = xyz[3*index+1];
		double z = xyz[3*index+2];

		double border;
		int curr_ind;
		double cur_two_power_x;

		int upto, diff;

		if(num_buckets == 1024){upto = 10; diff = 1073741823;}
		if(num_buckets == 512){upto = 9; diff   = 134217727;}
		if(num_buckets == 256){upto = 8; diff   = 16777215;}
		if(num_buckets == 128){upto = 7; diff   = 2097151;}
		if(num_buckets == 64){upto = 6; diff    = 262143;}
		if(num_buckets == 32){upto = 5; diff    = 32767;}
		if(num_buckets == 16){upto = 4; diff    = 4095;}
		///////////////
		border = 0;
		curr_ind = 0;
		cur_two_power_x = 0;
		//__syncthreads();
	
		cur_two_power_x = 1.0f;
		for(int kk=0;kk<upto;kk++)
		{
		cur_two_power_x = cur_two_power_x / 2.0f;
			if(x <= border)
			{	
			curr_ind = 2 * curr_ind + 1; 
			border = border - cur_two_power_x;
			}else
			{
			curr_ind = 2 * curr_ind + 2;
			border = border + cur_two_power_x;	
			};
		}
			
		border = 0.0f;
		cur_two_power_x = 1.0f;
		for(int kk=0;kk<upto;kk++)
		{
			cur_two_power_x = cur_two_power_x / 2.0f;	
			if(y <= border)
			{	
			curr_ind = 2 * curr_ind + 1; 
			border = border - cur_two_power_x;
			}else
			{
			curr_ind = 2 * curr_ind + 2;
			border = border + cur_two_power_x;	
			};
		}
	
	
		border = 0.0f;
		cur_two_power_x = 1.0f;
		for(int kk=0;kk<upto;kk++)
		{
			cur_two_power_x = cur_two_power_x / 2.0f;	
			if(z <= border)
			{	
			curr_ind = 2 * curr_ind + 1; 
			border = border - cur_two_power_x;
			}else
			{
			curr_ind = 2 * curr_ind + 2;
			border = border + cur_two_power_x;	
			};
		}
	
		curr_ind = curr_ind-diff;
		_d_index[index] = curr_ind;
	}
}

__global__ void kernel_CountPointsInBuckets(unsigned int *_d_index, int *_d_table_of_buckets, int *d_num_points_in_bucket, int numpoints_m)
{
	int index = getThreadId(blockIdx.x,blockIdx.y,blockIdx.z, threadIdx.x, threadIdx.y,threadIdx.z);
	int temp_index_of_bucket;
	int temp_previous_index_of_bucket;
	int amount_of_points = 0;

	if(index < numpoints_m)
	{
		if(index == 0)
		{
			amount_of_points = 1;
			temp_index_of_bucket = _d_index[0];
			for(int k = 1 ; k < numpoints_m; k++)
			{
				if(_d_index[index+k] == temp_index_of_bucket)
				{
					amount_of_points++;
				}
				else
					break;
			}
			_d_table_of_buckets[temp_index_of_bucket] = 0;
			d_num_points_in_bucket[temp_index_of_bucket] = amount_of_points;
		}
		else //index != 0
		{
			amount_of_points = 1;
			temp_index_of_bucket = _d_index[index];
			temp_previous_index_of_bucket = _d_index[index-1];
			if(temp_index_of_bucket != temp_previous_index_of_bucket)
			{
				_d_table_of_buckets[temp_index_of_bucket] = index;
				for(int k = 1 ; k < numpoints_m; k++)
				{
					if(index+k < numpoints_m)
					{
						if(_d_index[index+k] == temp_index_of_bucket)
						{
							amount_of_points++;
						}
						else
							break;
					}
					else
						break;
				}
			d_num_points_in_bucket[temp_index_of_bucket] = amount_of_points;	
			}
		}
		
	}
	
}

__global__ void kernel_FindNN(int *_d_NN, unsigned int *_d_index, int * d_num_points_in_bucket, 
					int * _d_table_of_buckets, unsigned int * _d_sorted_table_of_points, int numpoints_d, 
					int _thresholdOfPointsINNER, int _thresholdOfPointsOUTER, int num_buckets,
					double *_d_temp_distances, int _case, 
					double _additionalDistanceCheck,
					double *_xRef, double *_yRef, double *_zRef,
					double *_xToAlign, double *_yToAlign, double *_zToAlign, int repeat, int step)
{
	int index = getThreadId(blockIdx.x,blockIdx.y,blockIdx.z, threadIdx.x, threadIdx.y,threadIdx.z);

	if(index < numpoints_d)
	{
		int temp_curr_ind = _d_index[index];
		int curr_ind = 0;
		switch(_case)
		{
			case 0:	{curr_ind = temp_curr_ind - num_buckets * num_buckets - num_buckets - 1;	break;}
			case 1:	{curr_ind = temp_curr_ind - num_buckets - 1;	break;	}
			case 2:{curr_ind = temp_curr_ind + num_buckets * num_buckets - num_buckets - 1;	break;	}
			case 3:	{curr_ind = temp_curr_ind - num_buckets * num_buckets - 1;	break;	}
			case 4:	{curr_ind = temp_curr_ind - 1;	break;	}
			case 5:	{curr_ind = temp_curr_ind + num_buckets * num_buckets - 1;break;	}
			case 6:{curr_ind = temp_curr_ind - num_buckets * num_buckets + num_buckets - 1;	break;	}
			case 7:	{curr_ind = temp_curr_ind + num_buckets - 1;	break;	}
			case 8:	{curr_ind = temp_curr_ind + num_buckets * num_buckets + num_buckets - 1;	break;}
			case 9:	{curr_ind = temp_curr_ind - num_buckets * num_buckets - num_buckets;	break;	}
			case 10:{curr_ind = temp_curr_ind - num_buckets;break;	}
			case 11:{curr_ind = temp_curr_ind + num_buckets * num_buckets - num_buckets;	break;}
			case 12:{curr_ind = temp_curr_ind - num_buckets * num_buckets;	break;	}
			case 13:{curr_ind = temp_curr_ind;	break;	}
			case 14:{curr_ind = temp_curr_ind + num_buckets * num_buckets;break;	}
			case 15:{curr_ind = temp_curr_ind - num_buckets * num_buckets + num_buckets;	break;	}
			case 16:{curr_ind = temp_curr_ind + num_buckets;break;	}
			case 17:{curr_ind = temp_curr_ind + num_buckets * num_buckets + num_buckets;	break;	}
			case 18:{curr_ind = temp_curr_ind - num_buckets * num_buckets - num_buckets + 1;	break;	}
			case 19:{curr_ind = temp_curr_ind - num_buckets + 1;	break;	}
			case 20:{curr_ind = temp_curr_ind + num_buckets * num_buckets - num_buckets + 1;	break;	}
			case 21:{curr_ind = temp_curr_ind - num_buckets * num_buckets + 1;break;	}
			case 22:{curr_ind = temp_curr_ind + 1;	break;	}
			case 23:{curr_ind = temp_curr_ind + num_buckets * num_buckets+1;	break;	}
			case 24:{curr_ind = temp_curr_ind - num_buckets * num_buckets + num_buckets + 1;	break;	}
			case 25:{curr_ind = temp_curr_ind + num_buckets + 1;break;	}
			case 26:{curr_ind = temp_curr_ind + num_buckets * num_buckets + num_buckets + 1;	break;
			}
		}
		
		////////////////////////////////////d_compute_min_distances//////////////////////////////////////////////////////////////////////
		double d_min_distance = _d_temp_distances[index];
		double x_reference;
		double y_reference;
		double z_reference;

		double xRef = _xToAlign[index];
		double yRef = _yToAlign[index];
		double zRef = _zToAlign[index];

		//printf("A%d\n",index);

		int index_result = -1;
		int index_of_bucket = curr_ind;

		if(index_of_bucket<0)index_of_bucket = 0;
		if(index_of_bucket >= num_buckets*num_buckets*num_buckets) index_of_bucket = num_buckets*num_buckets*num_buckets-1;

		int number_of_points_in_bucket = d_num_points_in_bucket[index_of_bucket];//tu konflikt czytania
		int first_element_index = _d_table_of_buckets[index_of_bucket];				//tu konflikt czytania

		double temp_min_dist;

		if(_case == 13)
		{
			if(number_of_points_in_bucket > _thresholdOfPointsINNER)
				number_of_points_in_bucket = _thresholdOfPointsINNER;

		}else
		{
			if(number_of_points_in_bucket > _thresholdOfPointsOUTER)
				number_of_points_in_bucket = _thresholdOfPointsOUTER;
		}

		int endIndex = (repeat+1) * step;
		if(endIndex > number_of_points_in_bucket)
		{
			endIndex = number_of_points_in_bucket;
		}

		//printf("B%d  index_of_bucket = %d endIndex=%d\n",index,index_of_bucket,endIndex);

		for(int ii = repeat * step; ii < endIndex; ii++)
		{
			int index_of_point_in_sorted_table_of_points = _d_sorted_table_of_points[first_element_index + ii];//tu konflikt czytania

			x_reference = _xRef[index_of_point_in_sorted_table_of_points];
			y_reference = _yRef[index_of_point_in_sorted_table_of_points];
			z_reference = _zRef[index_of_point_in_sorted_table_of_points];


			temp_min_dist = sqrtf((xRef - x_reference)*(xRef -x_reference) +  
								  (yRef - y_reference)*(yRef -y_reference) + 
								  (zRef - z_reference)*(zRef -z_reference)); 

			if(temp_min_dist < d_min_distance) 
			{
				d_min_distance = temp_min_dist;
				index_result = index_of_point_in_sorted_table_of_points;
			}
		}
		//printf("C%d\n",index);

		if(index_result != -1)
		{
			if(d_min_distance < _additionalDistanceCheck )
			{
				_d_NN[index] = index_result;
				_d_temp_distances[index] = d_min_distance;
				//__syncthreads();
			}
		}
	}
}
__global__ void kernel_FindNN_ALL(int *_d_NN, unsigned int *_d_index, int * d_num_points_in_bucket, 
					int * _d_table_of_buckets, unsigned int * _d_sorted_table_of_points, int numpoints_d, 
					int num_buckets,
					double *_d_temp_distances, int _case, 
					double _additionalDistanceCheck,
					double *_xRef, double *_yRef, double *_zRef,
					double *_xToAlign, double *_yToAlign, double *_zToAlign)
{
	int index = getThreadId(blockIdx.x,blockIdx.y,blockIdx.z, threadIdx.x, threadIdx.y,threadIdx.z);

	if(index < numpoints_d)
	{
		int temp_curr_ind = _d_index[index];
		int curr_ind = 0;
		switch(_case)
		{
			case 0:	{curr_ind = temp_curr_ind - num_buckets * num_buckets - num_buckets - 1;	break;}
			case 1:	{curr_ind = temp_curr_ind - num_buckets - 1;	break;	}
			case 2:{curr_ind = temp_curr_ind + num_buckets * num_buckets - num_buckets - 1;	break;	}
			case 3:	{curr_ind = temp_curr_ind - num_buckets * num_buckets - 1;	break;	}
			case 4:	{curr_ind = temp_curr_ind - 1;	break;	}
			case 5:	{curr_ind = temp_curr_ind + num_buckets * num_buckets - 1;break;	}
			case 6:{curr_ind = temp_curr_ind - num_buckets * num_buckets + num_buckets - 1;	break;	}
			case 7:	{curr_ind = temp_curr_ind + num_buckets - 1;	break;	}
			case 8:	{curr_ind = temp_curr_ind + num_buckets * num_buckets + num_buckets - 1;	break;}
			case 9:	{curr_ind = temp_curr_ind - num_buckets * num_buckets - num_buckets;	break;	}
			case 10:{curr_ind = temp_curr_ind - num_buckets;break;	}
			case 11:{curr_ind = temp_curr_ind + num_buckets * num_buckets - num_buckets;	break;}
			case 12:{curr_ind = temp_curr_ind - num_buckets * num_buckets;	break;	}
			case 13:{curr_ind = temp_curr_ind;	break;	}
			case 14:{curr_ind = temp_curr_ind + num_buckets * num_buckets;break;	}
			case 15:{curr_ind = temp_curr_ind - num_buckets * num_buckets + num_buckets;	break;	}
			case 16:{curr_ind = temp_curr_ind + num_buckets;break;	}
			case 17:{curr_ind = temp_curr_ind + num_buckets * num_buckets + num_buckets;	break;	}
			case 18:{curr_ind = temp_curr_ind - num_buckets * num_buckets - num_buckets + 1;	break;	}
			case 19:{curr_ind = temp_curr_ind - num_buckets + 1;	break;	}
			case 20:{curr_ind = temp_curr_ind + num_buckets * num_buckets - num_buckets + 1;	break;	}
			case 21:{curr_ind = temp_curr_ind - num_buckets * num_buckets + 1;break;	}
			case 22:{curr_ind = temp_curr_ind + 1;	break;	}
			case 23:{curr_ind = temp_curr_ind + num_buckets * num_buckets+1;	break;	}
			case 24:{curr_ind = temp_curr_ind - num_buckets * num_buckets + num_buckets + 1;	break;	}
			case 25:{curr_ind = temp_curr_ind + num_buckets + 1;break;	}
			case 26:{curr_ind = temp_curr_ind + num_buckets * num_buckets + num_buckets + 1;	break;
			}
		}
		
		////////////////////////////////////d_compute_min_distances//////////////////////////////////////////////////////////////////////
		double d_min_distance = _d_temp_distances[index];
		double x_reference;
		double y_reference;
		double z_reference;

		double xRef = _xToAlign[index];
		double yRef = _yToAlign[index];
		double zRef = _zToAlign[index];

		//printf("A%d\n",index);

		int index_result = -1;
		int index_of_bucket = curr_ind;

		if(index_of_bucket<0)index_of_bucket = 0;
		if(index_of_bucket >= num_buckets*num_buckets*num_buckets) index_of_bucket = num_buckets*num_buckets*num_buckets-1;

		int number_of_points_in_bucket = d_num_points_in_bucket[index_of_bucket];//tu konflikt czytania
		int first_element_index = _d_table_of_buckets[index_of_bucket];				//tu konflikt czytania

		double temp_min_dist;

		

		int endIndex = number_of_points_in_bucket;
		

		//printf("B%d  index_of_bucket = %d endIndex=%d\n",index,index_of_bucket,endIndex);

		for(int ii = 0; ii < endIndex; ii++)
		{
			int index_of_point_in_sorted_table_of_points = _d_sorted_table_of_points[first_element_index + ii];//tu konflikt czytania

			x_reference = _xRef[index_of_point_in_sorted_table_of_points];
			y_reference = _yRef[index_of_point_in_sorted_table_of_points];
			z_reference = _zRef[index_of_point_in_sorted_table_of_points];


			temp_min_dist = sqrtf((xRef - x_reference)*(xRef -x_reference) +  
								  (yRef - y_reference)*(yRef -y_reference) + 
								  (zRef - z_reference)*(zRef -z_reference)); 

			if(temp_min_dist < d_min_distance) 
			{
				d_min_distance = temp_min_dist;
				index_result = index_of_point_in_sorted_table_of_points;
			}
		}
		//printf("C%d\n",index);

		if(index_result != -1)
		{
			if(d_min_distance < _additionalDistanceCheck )
			{
				_d_NN[index] = index_result;
				_d_temp_distances[index] = d_min_distance;
				//__syncthreads();
			}
		}
	}
	
}
__global__ void kernel_FindNN_ALL(int *_d_NN, unsigned int *_d_index, int * d_num_points_in_bucket, 
					int * _d_table_of_buckets, unsigned int * _d_sorted_table_of_points, int numpoints_d, 
					int num_buckets,
					double *_d_temp_distances, int _case, 
					double _additionalDistanceCheck,
					double *mxyz,
					double *dxyz)
{
	int index = getThreadId(blockIdx.x,blockIdx.y,blockIdx.z, threadIdx.x, threadIdx.y,threadIdx.z);

	if(index < numpoints_d)
	{
		int temp_curr_ind = _d_index[index];
		int curr_ind = 0;
		switch(_case)
		{
			case 0:	{curr_ind = temp_curr_ind - num_buckets * num_buckets - num_buckets - 1;	break;}
			case 1:	{curr_ind = temp_curr_ind - num_buckets - 1;	break;	}
			case 2:{curr_ind = temp_curr_ind + num_buckets * num_buckets - num_buckets - 1;	break;	}
			case 3:	{curr_ind = temp_curr_ind - num_buckets * num_buckets - 1;	break;	}
			case 4:	{curr_ind = temp_curr_ind - 1;	break;	}
			case 5:	{curr_ind = temp_curr_ind + num_buckets * num_buckets - 1;break;	}
			case 6:{curr_ind = temp_curr_ind - num_buckets * num_buckets + num_buckets - 1;	break;	}
			case 7:	{curr_ind = temp_curr_ind + num_buckets - 1;	break;	}
			case 8:	{curr_ind = temp_curr_ind + num_buckets * num_buckets + num_buckets - 1;	break;}
			case 9:	{curr_ind = temp_curr_ind - num_buckets * num_buckets - num_buckets;	break;	}
			case 10:{curr_ind = temp_curr_ind - num_buckets;break;	}
			case 11:{curr_ind = temp_curr_ind + num_buckets * num_buckets - num_buckets;	break;}
			case 12:{curr_ind = temp_curr_ind - num_buckets * num_buckets;	break;	}
			case 13:{curr_ind = temp_curr_ind;	break;	}
			case 14:{curr_ind = temp_curr_ind + num_buckets * num_buckets;break;	}
			case 15:{curr_ind = temp_curr_ind - num_buckets * num_buckets + num_buckets;	break;	}
			case 16:{curr_ind = temp_curr_ind + num_buckets;break;	}
			case 17:{curr_ind = temp_curr_ind + num_buckets * num_buckets + num_buckets;	break;	}
			case 18:{curr_ind = temp_curr_ind - num_buckets * num_buckets - num_buckets + 1;	break;	}
			case 19:{curr_ind = temp_curr_ind - num_buckets + 1;	break;	}
			case 20:{curr_ind = temp_curr_ind + num_buckets * num_buckets - num_buckets + 1;	break;	}
			case 21:{curr_ind = temp_curr_ind - num_buckets * num_buckets + 1;break;	}
			case 22:{curr_ind = temp_curr_ind + 1;	break;	}
			case 23:{curr_ind = temp_curr_ind + num_buckets * num_buckets+1;	break;	}
			case 24:{curr_ind = temp_curr_ind - num_buckets * num_buckets + num_buckets + 1;	break;	}
			case 25:{curr_ind = temp_curr_ind + num_buckets + 1;break;	}
			case 26:{curr_ind = temp_curr_ind + num_buckets * num_buckets + num_buckets + 1;	break;
			}
		}
		
		////////////////////////////////////d_compute_min_distances//////////////////////////////////////////////////////////////////////
		double d_min_distance = _d_temp_distances[index];
		double x_reference;
		double y_reference;
		double z_reference;

		double xRef = dxyz[3*index+0];
		double yRef = dxyz[3*index+1];
		double zRef = dxyz[3*index+2];

		//printf("A%d\n",index);

		int index_result = -1;
		int index_of_bucket = curr_ind;

		if(index_of_bucket<0)index_of_bucket = 0;
		if(index_of_bucket >= num_buckets*num_buckets*num_buckets) index_of_bucket = num_buckets*num_buckets*num_buckets-1;

		int number_of_points_in_bucket = d_num_points_in_bucket[index_of_bucket];//tu konflikt czytania
		int first_element_index = _d_table_of_buckets[index_of_bucket];				//tu konflikt czytania

		double temp_min_dist;

		

		int endIndex = number_of_points_in_bucket;
		

		//printf("B%d  index_of_bucket = %d endIndex=%d\n",index,index_of_bucket,endIndex);

		for(int ii = 0; ii < endIndex; ii++)
		{
			int index_of_point_in_sorted_table_of_points = _d_sorted_table_of_points[first_element_index + ii];//tu konflikt czytania

			x_reference = mxyz[3*index_of_point_in_sorted_table_of_points+0];
			y_reference = mxyz[3*index_of_point_in_sorted_table_of_points+1];
			z_reference = mxyz[3*index_of_point_in_sorted_table_of_points+2];


			temp_min_dist = sqrtf((xRef - x_reference)*(xRef -x_reference) +  
								  (yRef - y_reference)*(yRef -y_reference) + 
								  (zRef - z_reference)*(zRef -z_reference)); 

			if(temp_min_dist < d_min_distance) 
			{
				d_min_distance = temp_min_dist;
				index_result = index_of_point_in_sorted_table_of_points;
			}
		}
		//printf("C%d\n",index);

		if(index_result != -1)
		{
			if(d_min_distance < _additionalDistanceCheck )
			{
				_d_NN[index] = index_result;
				_d_temp_distances[index] = d_min_distance;
				//__syncthreads();
			}
		}
	}
	
}
void cudaFindNN(double *d_mxyz, 
				double *d_dxyz,
				unsigned int *index_m, 
				unsigned int *index_d, 
				unsigned int *_d_keysReference, 
				int *_d_table_of_buckets, 
				int *d_num_points_in_bucket, 
				int *_d_NN,
				int numpoints_m, 
				int numpoints_d, 
				int _thresholdOfPointsINNER, 
				int _thresholdOfPointsOUTER, 
				int num_buckets, 
				double *_d_temp_distances, 
				double _additionalDistanceCheck)
{
	CUDA_CHECK_ERROR();
	size_t mfree,mtot;
	dim3 block_m, block_d;
	//dim3 blockToFill;
	dim3 thread_m, thread_d;
	//dim3 threadToFill;
	
	getKernelDim(block_m, thread_m , numpoints_m);
	getKernelDim(block_d, thread_d , numpoints_d);
	//getKernelDimBuckets(blockToFill, threadToFill, num_buckets);
	
	printf("block_m: %d %d %d   thread_m: %d %d %d\n",block_m.x,block_m.y,block_m.z,thread_m.x,thread_m.y,thread_m.z);
	printf("block_d: %d %d %d   thread_d: %d %d %d\n",block_d.x,block_d.y,block_d.z,thread_d.x,thread_d.y,thread_d.z);
	
	hipMemGetInfo(&mfree,&mtot);
	printf("Now %lf MB is free from %lf MB total\n",mfree/(1024*1024.0),mtot/(1024*1024.0));
	
	CUDA_CHECK_ERROR();
	//Indexes for M
	kernel_ComputeIndexes<<<block_m, thread_m>>>(index_m, d_mxyz, numpoints_m, num_buckets);
	hipDeviceSynchronize();
	hipMemGetInfo(&mfree,&mtot);
	printf("Now %lf MB is free from %lf MB total\n",mfree/(1024*1024.0),mtot/(1024*1024.0));
	//kernel_initKey<<<block_m, thread_m>>>(_d_keysReference, numpoints_m);  //use sequence instead
	/*printf("--------sequence-----_d_keysReference-------\n");
	
	for(int i=0;i<numpoints_m;++i)
	{
		printf("_d_keysReference[%d] = %d\n",i,_d_keysReference[i]);
	}
	printf("---------kernel_ComputeIndexes-----------\n");
	
	for(int i=0;i<numpoints_m;++i)
	{
		printf("index_m[%d] = %d\n",i,index_m[i]);
	}printf("--------sequence-----_d_keysReference-------\n");
	
	for(int i=0;i<numpoints_m;++i)
	{
		printf("_d_keysReference[%d] = %d\n",i,_d_keysReference[i]);
	}
		*/
	CUDA_CHECK_ERROR();
	thrust::device_ptr <unsigned int> dev_ptrindex_m ( index_m );
	CUDA_CHECK_ERROR();
	thrust::device_ptr <unsigned int> dev_ptr_d_keysReference ( _d_keysReference );
	CUDA_CHECK_ERROR();
	/*printf("--------sequence-----_d_keysReference-------\n");
	
	for(int i=0;i<numpoints_m;++i)
	{
		printf("_d_keysReference[%d] = %d\n",i,_d_keysReference[i]);
	}*/
	thrust::sequence(dev_ptr_d_keysReference,dev_ptr_d_keysReference+numpoints_m);
	CUDA_CHECK_ERROR();
	
	hipDeviceSynchronize();
	/*printf("--------sequence-----_d_keysReference-------\n");
	
	for(int i=0;i<numpoints_m;++i)
	{
		printf("_d_keysReference[%d] = %d\n",i,_d_keysReference[i]);
	}*/
	thrust::sort_by_key (dev_ptrindex_m , dev_ptrindex_m + numpoints_m, dev_ptr_d_keysReference );
	CUDA_CHECK_ERROR();
	hipDeviceSynchronize();
	/*printf("----------sort_by_key----------\n");
	
	for(int i=0;i<numpoints_m;++i)
	{
		printf("index_m[%d] = %d\n",i,index_m[i]);
	}
	for(int i=0;i<numpoints_m;++i)
	{
		printf("_d_keysReference[%d] = %d\n",i,_d_keysReference[i]);
	}
	
	
	printf("\n\n\n\n");
	*/
	
	thrust::device_ptr <int> dev_ptr_d_table_of_buckets(_d_table_of_buckets);
	CUDA_CHECK_ERROR();
	thrust::device_ptr <int> dev_ptrd_num_points_in_bucket(d_num_points_in_bucket);
	CUDA_CHECK_ERROR();
	thrust::fill(dev_ptr_d_table_of_buckets,dev_ptr_d_table_of_buckets+num_buckets*num_buckets*num_buckets,-1);
	CUDA_CHECK_ERROR();
	thrust::fill(dev_ptrd_num_points_in_bucket,dev_ptrd_num_points_in_bucket+num_buckets*num_buckets*num_buckets,-1);
	
	//kernerl_Fill<<<blockToFill, threadToFill>>>(_d_table_of_buckets, num_buckets, -1);
	//kernerl_Fill<<<blockToFill, threadToFill>>>(d_num_points_in_bucket, num_buckets, -1);
	
	hipDeviceSynchronize();
	CUDA_CHECK_ERROR();
	hipMemGetInfo(&mfree,&mtot);
	printf("Now %lf MB is free from %lf MB total\n",mfree/(1024*1024.0),mtot/(1024*1024.0));
	/*
	printf("----------_d_table_of_buckets----------\n");
	
	for(int i=0;i<num_buckets*num_buckets*num_buckets;++i)
	{
		printf("_d_table_of_buckets[%d] = %d\n",i,_d_table_of_buckets[i]);
	}
	
	printf("----------d_num_points_in_bucket----------\n");
	
	for(int i=0;i<num_buckets*num_buckets*num_buckets;++i)
	{
		printf("d_num_points_in_bucket[%d] = %d\n",i,d_num_points_in_bucket[i]);
	}
	 */
	
	hipDeviceSynchronize();
	kernel_CountPointsInBuckets<<<block_m, thread_m>>>(index_m, _d_table_of_buckets, d_num_points_in_bucket, numpoints_m);
	hipDeviceSynchronize();
	CUDA_CHECK_ERROR();
		/*
	thrust::device_ptr <double> dev_ptr_d_temp_double_mem ( _d_temp_double_mem );
	thrust::device_ptr <double> dev_ptr_d_table_m_d ( _d_table_m_d );
	thrust::device_ptr <double> dev_ptr_d_table_C ( _d_table_C );
*/
	


	//Indexes for D
	kernel_ComputeIndexes<<<block_d, thread_d>>>(index_d, d_dxyz, numpoints_d, num_buckets);
	hipDeviceSynchronize();
	
	//kernel_fill_NN_by_value<<<block_d, thread_d>>>(_d_NN, numpoints_d, -1);	
	//hipDeviceSynchronize();
	thrust::device_ptr <int> dev_ptrd__d_NN(_d_NN);
	thrust::fill(dev_ptrd__d_NN,dev_ptrd__d_NN+numpoints_d,-1);
	
	CUDA_CHECK_ERROR();
	
	
	hipMemGetInfo(&mfree,&mtot);
	printf("Now %lf MB is free from %lf MB total\n",mfree/(1024*1024.0),mtot/(1024*1024.0));
	
	//kernel_fill_temp_distances_by_value<<<block_d, thread_d>>>(_d_temp_distances, numpoints_d, 2.0f);	
	//hipDeviceSynchronize();
	thrust::device_ptr <double> dev_ptrd__d_temp_distances(_d_temp_distances);
	thrust::fill(dev_ptrd__d_temp_distances,dev_ptrd__d_temp_distances+numpoints_d,2);


		CUDA_CHECK_ERROR();
/*
	int step = 10;
	int iter13 ;

	for(int kkk = 0; kkk < 27; kkk++)
	{
		if(kkk == 13)
		{
			iter13 = _thresholdOfPointsINNER/step + 1;
		}
		else
		{
			iter13 = _thresholdOfPointsOUTER/step + 1;
		}

		for(int kk = 0; kk < iter13; kk++)
		{
			kernel_FindNN<<<block_d, thread_d>>>(_d_NN, index_d, d_num_points_in_bucket, 
					_d_table_of_buckets, _d_keysReference, numpoints_d, 
					_thresholdOfPointsINNER, _thresholdOfPointsOUTER, num_buckets,
					_d_temp_distances, kkk, 
					_additionalDistanceCheck,
					d_mx, d_my, d_mz, 
					d_dx,d_dy, d_dz,kk,step);
			hipDeviceSynchronize();
		CUDA_CHECK_ERROR();
		}
	}*/
		
		
	
	for(int kkk = 0; kkk < 27; kkk++)
	{

		kernel_FindNN_ALL<<<block_d, thread_d>>>(_d_NN, index_d, d_num_points_in_bucket, 
				_d_table_of_buckets, _d_keysReference, numpoints_d, 
				num_buckets,
				_d_temp_distances, kkk, 
				_additionalDistanceCheck,
				d_mxyz, 
				d_dxyz);
		hipDeviceSynchronize();
		CUDA_CHECK_ERROR();
	}

	
	hipMemGetInfo(&mfree,&mtot);
	printf("Now %lf MB is free from %lf MB total\n",mfree/(1024*1024.0),mtot/(1024*1024.0));

}

/*
__global__ void kernel_bbox(double *d_x, double *d_y, double *d_z, 
							double *bbox, unsigned int size)
{
	int index = getThreadId(blockIdx.x,blockIdx.y,blockIdx.z, threadIdx.x, threadIdx.y,threadIdx.z);
	
	//Initialize the bbox with max values
	if(index==0)
	{
		bbox[0]=DBL_MAX;	//Xmin
		bbox[1]=-DBL_MAX;	//Xmax
		
		bbox[2]=DBL_MAX;	//Ymin
		bbox[3]=-DBL_MAX;	//Ymax
		
		bbox[4]=DBL_MAX;	//Zmin
		bbox[5]=-DBL_MAX;	//Zmax
	}
	
	__syncthreads();
	
	if(index<size)
	{
		
	}
}*/


void FindBoundingBox(double *d_x, double *d_y, double *d_z, double *bbox, unsigned int size)
{
	/*
	dim3 blocks, threads;
	getKernelDim(blocks, threads , size);
	
	kernel_bbox<<<blocks,threads>>>(d_x,d_y,d_z,size);
	*/
	
	// X       Y       Z
	bbox[0]=bbox[2]=bbox[4]=DBL_MAX;	//MIN
	bbox[1]=bbox[3]=bbox[5]=-DBL_MAX;	//MAX

	for(unsigned int i=0;i<size;i++)
	{
		if(d_x[i]<bbox[0])
			bbox[0]=d_x[i];
		if(d_x[i]>bbox[1])
			bbox[1]=d_x[i];

		if(d_y[i]<bbox[2])
			bbox[2]=d_y[i];
		if(d_y[i]>bbox[3])
			bbox[3]=d_y[i];

		if(d_z[i]<bbox[4])
			bbox[4]=d_z[i];
		if(d_z[i]>bbox[5])
			bbox[5]=d_z[i];
	}		
}