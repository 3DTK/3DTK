#include "hip/hip_runtime.h"
/** @file
 *  @brief GPU-ICP Algorithm
 *  @author Deyuan Qiu, University of Applied Sciences Bonn-Rhein-Sieg, Sankt Augustin, Germany.
 *                      Fraunhofer IAIS, Sankt Augustin, Germany.
 */


#include "CIcpGpuCuda_kernel.cuh"
#include "CIcpGpuCuda.cuh"
#include "CSystem.h"

void CIcpGpuCuda::init(unsigned unWidth, unsigned unHeight, unsigned max_iter){

	// Initialize CUTIL
	int d;
//	CUT_DEVICE_INIT(_argc, _argv);
	hipSetDevice(0); // Since we have only one GPU I didn't initialize anything
	// It may cause problems if more GPUs take into account
	// The reason of doing that was initialization of it several times
	// Now it is not initialized several times and just once.

        unMaxIteration = max_iter;
        matrices = (Matrix**)malloc(sizeof(Matrix*) * max_iter);
        for(int i = 0 ; i < max_iter ; ++i){
            matrices[i] = new Matrix(4,4);
            Matrix* m = matrices[i];
            (*m)(1,1) = 1;(*m)(1,2) = 0;(*m)(1,3) = 0;(*m)(1,4) = 0;
            (*m)(2,1) = 0;(*m)(2,2) = 1;(*m)(2,3) = 0;(*m)(2,4) = 0;
            (*m)(3,1) = 0;(*m)(3,2) = 0;(*m)(3,3) = 1;(*m)(3,4) = 0;
            (*m)(4,1) = 0;(*m)(4,2) = 0;(*m)(4,3) = 0;(*m)(4,4) = 1;
        }

	// init by myself for dubugging
//    hipDeviceProp_t deviceProp;
//    deviceProp.major = 1;
//    deviceProp.minor = 0;
//    int desiredMinorRevision = 0;
//    int dev;
//
//    CUDA_SAFE_CALL(hipChooseDevice(&dev, &deviceProp));
//    CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, dev));
//
//    if(deviceProp.major > 1 || deviceProp.minor >= desiredMinorRevision)
//    {
//        printf("Using Device %d: \"%s\"\n", dev, deviceProp.name);
//        CUDA_SAFE_CALL(hipSetDevice(dev));
//    }
//    else if (desiredMinorRevision == 3)
//    {
//        printf("There is no device supporting compute capability %d.%d.\n\n",
//            1, desiredMinorRevision);
//        printf("TEST PASSED");
//        CUT_EXIT(_argc, _argv);
//    }


    //set data size
    setResolution(unWidth, unHeight);
//    cout<<"unSizeData: "<<unSizeData<<endl;
//    cout<<"Tree Size :"	<<TREESIZE<<endl;

    // Initialize CUBLAS
    hipblasStatus_t statusCUBLAS = cublasInit();
    if (statusCUBLAS != HIPBLAS_STATUS_SUCCESS) {
        cout<<"The error status is \n";
        cout<<statusCUBLAS<<endl;
        fprintf (stderr, "!!!! CUBLAS initialization error\n");
        exit(1);
    }

    // Initialize CUDPP
    CUDPPConfiguration config;
    config.datatype = CUDPP_FLOAT;
    config.algorithm = CUDPP_COMPACT;
    config.options = CUDPP_OPTION_FORWARD;
    result = cudppPlan(&compactplan, config, unSizeData, 1, 0);
    if (CUDPP_SUCCESS != result)	printf("Error creating CUDPPPlan\n");

	_unSizeTree = (unsigned)TREESIZE;
	CUDA_SAFE_CALL(hipHostMalloc((void**)&fSplit, _unSizeTree*sizeof(float)));
	CUDA_SAFE_CALL(hipHostMalloc((void**)&unIdx, _unSizeTree*sizeof(unsigned)));
	CUDA_SAFE_CALL(hipHostMalloc((void**)&unAxis, _unSizeTree*sizeof(unsigned)));
	CUDA_SAFE_CALL(hipHostMalloc((void**)&bIsLeaf, _unSizeTree*sizeof(bool)));
	CUDA_SAFE_CALL(hipHostMalloc((void**)&fLoBound, _unSizeTree*sizeof(float)));
	CUDA_SAFE_CALL(hipHostMalloc((void**)&fHiBound, _unSizeTree*sizeof(float)));

	CUDA_SAFE_CALL(hipHostMalloc((void**)&f4Mdl, unSizeData*sizeof(float4)));		//to be downloaded to texture

	// Host memory allocation
	CUDA_SAFE_CALL(hipHostMalloc((void**)&fHstScnX, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipHostMalloc((void**)&fHstScnY, unSizeData *sizeof(float)));
	CUDA_SAFE_CALL(hipHostMalloc((void**)&fHstScnZ, unSizeData*sizeof(float)));	//scene

	fHstScn[0]=fHstScnX;
	fHstScn[1]=fHstScnY;
	fHstScn[2]=fHstScnZ;

	CUDA_SAFE_CALL(hipHostMalloc((void**)&pNoPairs, sizeof(unsigned)));

	CSystem<double>::allocate(unSizeData,3,h_idata);							//model


/*
	h_idata = (double**)malloc(unSizeData*sizeof(double*));
	for (int unRow = 1 ; unRow < unSizeData ; unRow++)
	{
		h_idata[unRow] = (double*)malloc(3*sizeof(double));
	}
*/
	// Device memory allocation
	CUDA_SAFE_CALL(hipMalloc((void**)&fDist, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fDistCpt, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&unMask, unSizeData*sizeof(unsigned)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fDevMdlPairX, unSizeData*sizeof(float)));	//pairs after shrinking
	CUDA_SAFE_CALL(hipMalloc((void**)&fDevMdlPairY, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fDevMdlPairZ, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fDevScnPairX, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fDevScnPairY, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fDevScnPairZ, unSizeData*sizeof(float)));
        /////////////// Added by Shams
        CUDA_SAFE_CALL(hipMalloc((void**)&cngfDevScnX,unSizeData*sizeof(float)));
        CUDA_SAFE_CALL(hipMalloc((void**)&cngfDevScnY,unSizeData*sizeof(float)));
        CUDA_SAFE_CALL(hipMalloc((void**)&cngfDevScnZ,unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&cngfDevMdlPairX, unSizeData*sizeof(float)));	//pairs after shrinking
	CUDA_SAFE_CALL(hipMalloc((void**)&cngfDevMdlPairY, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&cngfDevMdlPairZ, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&cngfDevScnPairX, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&cngfDevScnPairY, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&cngfDevScnPairZ, unSizeData*sizeof(float)));

        ///////////////
	CUDA_SAFE_CALL(hipMalloc((void**)&fCenModX, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fCenModY, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fCenModZ, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fCenScnX, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fCenScnY, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fCenScnZ, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&unNoPairs, sizeof(unsigned)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fDevScnX, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fDevScnY, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fDevScnZ, unSizeData*sizeof(float)));

	CUDA_SAFE_CALL(hipMalloc((void**)&fDevSplit, _unSizeTree*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&unDevIdx, _unSizeTree*sizeof(unsigned)));
	CUDA_SAFE_CALL(hipMalloc((void**)&unDevAxis, _unSizeTree*sizeof(unsigned)));
	CUDA_SAFE_CALL(hipMalloc((void**)&bDevIsLeaf, _unSizeTree*sizeof(bool)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fDevLoBound, _unSizeTree*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fDevHiBound, _unSizeTree*sizeof(float)));


	CUDA_SAFE_CALL(hipMallocArray(&cuArray, &cuDesc, _unWidth, _unHeight));		//to be bound to texture

	// Initialize states
//	unMaxIteration		=	0;
	fMaxProcTime		=	0.0f;
	fMaxDeviation		=	0.0f;
	_fSearchRadiusMax	=	0.0f;
	_fSearchRadiusMin	=	0.0f;
	_fRadiusStep		=	0.0f;
	_unNoQSizeStep		=	0;
	_dElapsedTime		=	0.0;

        /*
            Array of ones to be used instead of abs sum
        */
/*
///////// From Teo Test
 float * d_a,*h_a,*h_b;
 int N = 10000;
  h_a=(float *)malloc(N*sizeof(float));
  h_b=(float *)malloc(N*sizeof(float));
  for (int i=0;i<N;i++)
    h_a[i]=1.0;

  hipMalloc((void**)&d_a,N*sizeof(float));
  hipMemcpy(d_a,h_a,N*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(h_b,d_a,N*sizeof(float),hipMemcpyDeviceToHost);
  printf(" dummy value %f = %f \n ",h_a[0],h_b[0]);
  hipFree(d_a);
  free(h_a);
  free(h_b);

////////
*/

	hipHostMalloc((void**)&temp_ones, unSizeData*sizeof(float));
	hipMalloc((void**)&ones, unSizeData*sizeof(float));	//Array of ones

        for(int i = 0; i < unSizeData ; ++i)
            temp_ones[i] = 1.0f;
	hipMemcpy(ones, temp_ones, unSizeData*sizeof(float), hipMemcpyHostToDevice);
}

CIcpGpuCuda::~CIcpGpuCuda(){
    /////////////
	// tidy up
    /////////////
//	CUDA_SAFE_CALL(hipFree(fDevMdlX));
//	CUDA_SAFE_CALL(hipFree(fDevMdlY));
//	CUDA_SAFE_CALL(hipFree(fDevMdlZ));
	CUDA_SAFE_CALL(hipUnbindTexture(refTex));
	CUDA_SAFE_CALL(hipFreeArray(cuArray));
	CUDA_SAFE_CALL(hipFree(fDevSplit));
	CUDA_SAFE_CALL(hipFree(unDevIdx));
	CUDA_SAFE_CALL(hipFree(unDevAxis));
	CUDA_SAFE_CALL(hipFree(bDevIsLeaf));
	CUDA_SAFE_CALL(hipFree(fDevLoBound));
	CUDA_SAFE_CALL(hipFree(fDevHiBound));
//	CUDA_SAFE_CALL(hipFree(unDevResult));
//	CUDA_SAFE_CALL(hipFree(temp));
	CUDA_SAFE_CALL(hipFree(fDevScnX));
	CUDA_SAFE_CALL(hipFree(fDevScnY));
	CUDA_SAFE_CALL(hipFree(fDevScnZ));
	CUDA_SAFE_CALL(hipFree(fDist));
	CUDA_SAFE_CALL(hipFree(fDistCpt));
	CUDA_SAFE_CALL(hipFree(fCenModX));
	CUDA_SAFE_CALL(hipFree(fCenModY));
	CUDA_SAFE_CALL(hipFree(fCenModZ));
	CUDA_SAFE_CALL(hipFree(fCenScnX));
	CUDA_SAFE_CALL(hipFree(fCenScnY));
	CUDA_SAFE_CALL(hipFree(fCenScnZ));
	CUDA_SAFE_CALL(hipFree(unMask));
	CUDA_SAFE_CALL(hipFree(fDevMdlPairX));
	CUDA_SAFE_CALL(hipFree(fDevMdlPairY));
	CUDA_SAFE_CALL(hipFree(fDevMdlPairZ));
	CUDA_SAFE_CALL(hipFree(fDevScnPairX));
	CUDA_SAFE_CALL(hipFree(fDevScnPairY));
	CUDA_SAFE_CALL(hipFree(fDevScnPairZ));
        ///// Added By Shams
        CUDA_SAFE_CALL(hipFree(cngfDevScnX));
        CUDA_SAFE_CALL(hipFree(cngfDevScnY));
        CUDA_SAFE_CALL(hipFree(cngfDevScnZ));
	CUDA_SAFE_CALL(hipFree(cngfDevMdlPairX));
	CUDA_SAFE_CALL(hipFree(cngfDevMdlPairY));
	CUDA_SAFE_CALL(hipFree(cngfDevMdlPairZ));
	CUDA_SAFE_CALL(hipFree(cngfDevScnPairX));
	CUDA_SAFE_CALL(hipFree(cngfDevScnPairY));
	CUDA_SAFE_CALL(hipFree(cngfDevScnPairZ));
        /////

	CUDA_SAFE_CALL(hipFree(unNoPairs));
	CUDA_SAFE_CALL(hipFree(ones));


//	CUDA_SAFE_CALL(hipHostFree(fX));
//	CUDA_SAFE_CALL(hipHostFree(fY));
//	CUDA_SAFE_CALL(hipHostFree(fZ));
	CUDA_SAFE_CALL(hipHostFree(fSplit));
	CUDA_SAFE_CALL(hipHostFree(unIdx));
	CUDA_SAFE_CALL(hipHostFree(unAxis));
	CUDA_SAFE_CALL(hipHostFree(bIsLeaf));
	CUDA_SAFE_CALL(hipHostFree(fHstScnX));
	CUDA_SAFE_CALL(hipHostFree(fHstScnY));
	CUDA_SAFE_CALL(hipHostFree(fHstScnZ));
	CUDA_SAFE_CALL(hipHostFree(fLoBound));
	CUDA_SAFE_CALL(hipHostFree(fHiBound));
	CUDA_SAFE_CALL(hipHostFree(pNoPairs));
	CUDA_SAFE_CALL(hipHostFree(f4Mdl));
	CUDA_SAFE_CALL(hipHostFree(temp_ones));

//	delete[] fX;
//	delete[] fY;
//	delete[] fZ;
//	delete[] fSplit;
//	delete[] unIdx;
//	delete[] unAxis;
//	delete[] bIsLeaf;
//	delete[] fLoBound;
//	delete[] fHiBound;
//	delete pNoPairs;

//	delete[] fHstScnX;
//	delete[] fHstScnY;
//	delete[] fHstScnZ;

//    delete kdTree;
//    delete st;


	free(h_idata);


    /////////
    // Exit
    /////////
    bool bShutDownSuccess = true;

    //Done with CUDPP
    result = cudppDestroyPlan(compactplan);
    if (CUDPP_SUCCESS != result){
    	printf("Error destroying CUDPPPlan\n");
    	bShutDownSuccess = false;
    }
//    else	printf("\nExit CUDPP.\n");

    //Done with CUBLAS
    hipblasStatus_t statusCUBLAS = cublasShutdown();
    if (statusCUBLAS != HIPBLAS_STATUS_SUCCESS) {
        fprintf (stderr, "!!!! shutdown error (A)\n");
        bShutDownSuccess = false;
    }
//    else	printf("\nExit CUBLAS.\n");

    //Done with ANN
	annClose();
	printf("\nExit ANN.\n");

	//Done with CUTIL
//	CUT_EXIT(_argc, _argv);
//	CUT_EXIT();
//	printf("\nExit CUTIL.\n");

/*
	if(bShutDownSuccess){
		printf("\nICP done successfully.\n");
	}
	cout<< "Last line of destructor\n";
*/
}

void CIcpGpuCuda::setResolution(unsigned unWidth, unsigned unHeight){
	if (unWidth>0 && unHeight>0) {
		_unWidth = unWidth;
		_unHeight = unHeight;
		unSizeData = _unWidth*_unHeight;

		//configure block and grid size
		unsigned unBlockSize = (unsigned)BLOCKSIZE;	//192
		if(unSizeData<=64){
			unNoThreads = 64;
			unNoBlocks = 1;
		}
		else if(unSizeData<=128){
			unNoThreads = 128;
			unNoBlocks = 1;
		}
		else if(unSizeData<=unBlockSize){
			unNoThreads = unBlockSize;
			unNoBlocks = 1;
		}
		else{
			unNoThreads = unBlockSize;
			if(unSizeData%unBlockSize)	unNoBlocks = unSizeData/unBlockSize + 1;
			else	unNoBlocks = unSizeData/unBlockSize;
		}
		cout<<"unNoThreads: "<<unNoThreads<<'\t'<<"unNoBlocks: "<<unNoBlocks<<endl;
		dimBlock.x=unNoThreads;
		dimBlock.y=1;
		dimBlock.z=1;
		dimGrid.x=unNoBlocks;
		dimGrid.y=1;
		dimGrid.z=1;
	}
	else{
		cout<<"Each aspect of resolution must be bigger than 0."<<endl;
		exit(1);
	}
}

void CIcpGpuCuda::setMaxIteration(unsigned unTimes){
	if(unTimes>0)	unMaxIteration = unTimes;
	else{
		cout<<"Error setting maximum iterations."<<endl;
		exit(1);
	}
}

void CIcpGpuCuda::setMaxProcTime(double dMilliseconds){
	if(dMilliseconds>0)	fMaxProcTime = dMilliseconds;
	else{
		cout<<"Error setting maximum processing time."<<endl;
		exit(1);
	}
}

void CIcpGpuCuda::setMaxDeviation(double fDeviation){
	if(fDeviation>0)	fMaxDeviation = fDeviation;
	else{
		cout<<"Error setting maximum deviation."<<endl;
		exit(1);
	}
}

void CIcpGpuCuda::setSearchRadius(float fRadiusMax, float fRadiusMin, unsigned unIterations){
	if((fRadiusMax>=fRadiusMin)&&(unIterations>0)){
		_fSearchRadiusMax = fRadiusMax;
		_fSearchRadiusMin = fRadiusMin;
		_fRadiusStep = (_fSearchRadiusMax-_fSearchRadiusMin)/(float)unIterations;
		_unIterations = unIterations;
		_unNoQSizeStep = unIterations/(unsigned)NO_QSIZE;
	}
	else{
		cout<<"Error setting search radius."<<endl;
		exit(1);
	}
}

float** CIcpGpuCuda::getScenePointer() {
	return fHstScn;
}

double** CIcpGpuCuda::getModelPointer(void){
	return h_idata;
}

unsigned CIcpGpuCuda::getSize(void){
	return unSizeData;
}

void CIcpGpuCuda::setTreePointer(ANNkd_tree *&tree){
    kdTree = tree;
}
void CIcpGpuCuda::getTreePointer(ANNkd_tree *&tree ){
    tree = kdTree;
}


void CIcpGpuCuda::setTree(){
	//build tree
//	fair::CTimer treetimer;
//	treetimer.reset();
//	kdTree = new ANNkd_tree(h_idata,(int)unSizeData, 3, 1, ANN_KD_STD); //commented by shams
//	cout<<"tree built in: "<<treetimer.getTime()<<endl;

	//preparation
	st = new ANNkdStats();
	kdTree->getStats(*st);
	int nDepth = st->depth;
//	cout<<"level of the tree: "<<st->depth<<" (counted from 0)"<<endl;
	unSizeTree = depth2size(nDepth);	//cout<<"unSizeTree: "<<unSizeTree<<endl;	  //decide size of array to be uploaded to GPU

	if(unSizeTree>_unSizeTree){
		cout<<"Not enough memory for tree construction. Tree size must be larger than "<<unSizeTree<<endl;
		exit(1);
	}

	//rearrange
	ANNkd_split* pRoot = (ANNkd_split*)kdTree->getRoot();
	if(unSizeData>1)	rearrange(pRoot, 1);
	else{
		cout<<"Not enough points in the tree."<<endl;
		exit(1);
	}

	//download the tree
	CUDA_SAFE_CALL(hipMemcpy(fDevSplit, fSplit, unSizeTree*sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(unDevIdx, unIdx, unSizeTree*sizeof(unsigned), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(unDevAxis, unAxis, unSizeTree*sizeof(unsigned), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(bDevIsLeaf, bIsLeaf, unSizeTree*sizeof(bool), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(fDevLoBound, fLoBound, unSizeTree*sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(fDevHiBound, fHiBound, unSizeTree*sizeof(float), hipMemcpyHostToDevice));

	//clean up
	delete kdTree;
	delete st;
}

void CIcpGpuCuda::setModel(){

	//using linear memory
//	CUDA_SAFE_CALL(hipHostMalloc((void**)&fX, unSizeData*sizeof(float)));			//structure of array (SoA), page-locked memory
//	CUDA_SAFE_CALL(hipHostMalloc((void**)&fY, unSizeData*sizeof(float)));
//	CUDA_SAFE_CALL(hipHostMalloc((void**)&fZ, unSizeData*sizeof(float)));
//	CUDA_SAFE_CALL(hipMalloc((void**)&fDevMdlX, unSizeData*sizeof(float)));
//	CUDA_SAFE_CALL(hipMalloc((void**)&fDevMdlY, unSizeData*sizeof(float)));
//	CUDA_SAFE_CALL(hipMalloc((void**)&fDevMdlZ, unSizeData*sizeof(float)));
//	for(unsigned i=0;i<unSizeData;i++){
//		fX[i]=(float)h_idata[i][0];//	fHstScnX[i]=fX[i] + 0.1;
//		fY[i]=(float)h_idata[i][0];//	fHstScnY[i]=fY[i] + 0.08;
//		fZ[i]=(float)h_idata[i][0];//	fHstScnZ[i]=fZ[i] + 0.12;
//	}
//	CUDA_SAFE_CALL(hipMemcpy(fDevMdlX, fX, unSizeData*sizeof(float), hipMemcpyHostToDevice));
//	CUDA_SAFE_CALL(hipMemcpy(fDevMdlY, fY, unSizeData*sizeof(float), hipMemcpyHostToDevice));
//	CUDA_SAFE_CALL(hipMemcpy(fDevMdlZ, fZ, unSizeData*sizeof(float), hipMemcpyHostToDevice));

	//using texture memory
	for(unsigned i=0;i<unSizeData;i++){	//type cast
		f4Mdl[i].x=(float)h_idata[i][0];//cout<<f4Mdl[i].x<<'\t';
		f4Mdl[i].y=(float)h_idata[i][1];//cout<<f4Mdl[i].y<<'\t';
		f4Mdl[i].z=(float)h_idata[i][2];//cout<<f4Mdl[i].z<<endl;
	}
	hipMemcpyToArray(cuArray,0,0,f4Mdl,unSizeData*sizeof(float4),hipMemcpyHostToDevice);
	hipBindTextureToArray(refTex,cuArray);
}

void CIcpGpuCuda::setScene(){
	CUDA_SAFE_CALL(hipMemcpy(fDevScnX, fHstScnX, unSizeData*sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(fDevScnY, fHstScnY, unSizeData*sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(fDevScnZ, fHstScnZ, unSizeData*sizeof(float), hipMemcpyHostToDevice));
}

inline int CIcpGpuCuda::depth2size(int nDepth)
{
	double nSize = 0;
	for(int i=0;i<=nDepth;i++)
	{
		nSize += pow(2.0,(double)i);
	}
	return (int)(nSize+0.5);
}

void CIcpGpuCuda::rearrange(ANNkd_ptr root, unsigned unStart)
{
	if(root!=NULL && root!=KD_TRIVIAL){
		if (root->isLeaf()) {
			bIsLeaf[unStart-1] = true;
			unIdx[unStart-1] = (unsigned)(((ANNkd_leaf*)root)->getIdxArray())[0];
		}
		else {
			ANNkd_ptr nL = ((ANNkd_split*)root)->getLeftChild();
			ANNkd_ptr nR = ((ANNkd_split*)root)->getRightChild();

			bIsLeaf[unStart-1] = false;
			fSplit[unStart-1] = (float)((ANNkd_split*)root)->getCutVal();
			unAxis[unStart-1] = (unsigned)((ANNkd_split*)root)->getCutDim();
			fLoBound[unStart-1] = ((ANNkd_split*)root)->getLoBound();
			fHiBound[unStart-1] = ((ANNkd_split*)root)->getHiBound();
			rearrange(nL, unStart*2);
			rearrange(nR, unStart*2+1);
		}
	}
}

void CIcpGpuCuda::setPointClouds(void){
	setTree();
	setModel();
	setScene();
}

void CIcpGpuCuda::iteration(){

	//////////////////////////
	// prepare for iteration
	//////////////////////////

	EnumIcpState icpStat = ICP_PROCESSING;

	float fSearchRadius = _fSearchRadiusMax;
	unsigned unNoIter = 0;
	float fDeviation = 0.0;
	unsigned unQStep = 0;

	final_matrix = new Matrix(4,4);
	(*final_matrix)(1,1) = 1.0;(*final_matrix)(1,2) = 0.0;(*final_matrix)(1,3) = 0.0;(*final_matrix)(1,4) = 0.0;
	(*final_matrix)(2,1) = 0.0;(*final_matrix)(2,2) = 1.0;(*final_matrix)(2,3) = 0.0;(*final_matrix)(2,4) = 0.0;
	(*final_matrix)(3,1) = 0.0;(*final_matrix)(3,2) = 0.0;(*final_matrix)(3,3) = 1.0;(*final_matrix)(3,4) = 0.0;
	(*final_matrix)(4,1) = 0.0;(*final_matrix)(4,2) = 0.0;(*final_matrix)(4,3) = 0.0;(*final_matrix)(4,4) = 1.0;

	Matrix matrix(4,4);
	
	init_time = clock();


        //The main loop of ICP
	while(icpStat == ICP_PROCESSING){
            if(unNoIter<=_unIterations)
		unQStep = unNoIter/_unNoQSizeStep;

            findNearestNeighbors(fSearchRadius, unQStep);

            result=cudppCompact(compactplan, fDistCpt, (size_t*)unNoPairs,
                     fDist, unMask, (size_t)unSizeData);

	    if (CUDPP_SUCCESS != result)
                printf("Error cudppCompact\n");
            CUDA_SAFE_CALL(hipMemcpy(pNoPairs, unNoPairs, sizeof(unsigned), hipMemcpyDeviceToHost));
            unPairs = *pNoPairs;

            if (unPairs) {
		//////////////////////
		// transform estimation
		//////////////////////

        	//Compute centroids (assume all data are non-negative)
		float *fCm = new float(3);
		float *fCs = new float(3);


                computeCentroid(fDevMdlPairX,fDevMdlPairY,fDevMdlPairZ, fCm);
                computeCentroid(fDevScnPairX,fDevScnPairY,fDevScnPairZ, fCs);

//                cout<<"Model Centroid "<<fCm[0]<<" "<<fCm[1]<<" "<<fCm[2]<<endl;
//                cout<<"Scene Centroid "<<fCs[0]<<" "<<fCs[1]<<" "<<fCs[3]<<endl;

                fDeviation = hipblasSdot(unSizeData,fDistCpt,1,ones,1);
                fDeviation /= unPairs;
                getCublasErr();

		// check for termination conditions
		unNoIter++;
		if(unNoIter<_unIterations)	
			fSearchRadius-=_fRadiusStep;
		if(fDeviation <= fMaxDeviation)
			icpStat = ICP_SUCCESS;
		else if(unNoIter >= unMaxIteration + 1) // unNoIter starts from 1
			icpStat = ICP_MAXITERATIONS;
		else if ( (double)(clock() - init_time)/ (double)CLOCKS_PER_SEC * 1000>= fMaxProcTime )
			icpStat = ICP_TIMEELAPSED;
		else
		{
			//Calculate centered point pairs
			class_centralize(unMask,
					fDevMdlPairX,fDevMdlPairY,fDevMdlPairZ,fDevScnPairX,fDevScnPairY,fDevScnPairZ,
					fCm[0],fCm[1],fCm[2],fCs[0],fCs[1],fCs[2],
					fCenModX,fCenModY,fCenModZ,fCenScnX,fCenScnY,fCenScnZ);

			//Fill H matrix
                        Matrix H(3,3);
                        H = computeHMatrix();

			//SVD
			Matrix U(3,3);
			DiagonalMatrix Lamda(3);
			Matrix V(3,3);
			SVD(H,Lamda,U,V);

			//Get rotation
			Matrix R(3,3);
			R = V*(U.t());

			// Calculate translation
			double dTranslation[3];
			ColumnVector col_vec(3);
			for(unsigned j = 0; j < 3; j++)
				col_vec(j+1) = fCs[j];

			ColumnVector r_time_colVec = ColumnVector(R*col_vec);

			dTranslation[0] = fCm[0] - r_time_colVec(1);
			dTranslation[1] = fCm[1] - r_time_colVec(2);
			dTranslation[2] = fCm[2] - r_time_colVec(3);

                        matrix = fillHomoMatrix(&R,dTranslation);

			*final_matrix = matrix * (*final_matrix);

                        for(int i = 1 ; i < 5 ; ++i)
                            for(int j = 1; j < 5 ; ++j){
                                (*matrices[unNoIter - 1])(i,j) = (matrix)(i,j);
                            }

			/////////////
			// transform
			/////////////

			class_transformation(fDevScnX, fDevScnY, fDevScnZ,
					(float)matrix(1,1),	(float)matrix(1,2),	(float)matrix(1,3),	(float)matrix(1,4),
					(float)matrix(2,1),	(float)matrix(2,2),	(float)matrix(2,3),	(float)matrix(2,4),
					(float)matrix(3,1),	(float)matrix(3,2),	(float)matrix(3,3),	(float)matrix(3,4));


			///////////////
			// termination
			///////////////

			}
		}//if(unPairs)
		else	icpStat = ICP_NOTMATCHABLE;
	}//while(icpStat == ICP_PROCESSING)

	_dElapsedTime = (double)(clock() - init_time)/(double)CLOCKS_PER_SEC * 1000.0;		//temporary
	cout<<"=========="<<endl;
	switch(icpStat)
	{
		case	ICP_LIMIT:	
        		cout<<"terminated: convergent limit reached."<<endl;
                        break;
		case	ICP_NOTMATCHABLE:	
                        cout<<"terminated: point clounds not matchable."<<endl;
                        break;
		case	ICP_MAXITERATIONS:	
                        cout<<"terminated: maximum iteration exceeds."<<endl;
                        break;
		case	ICP_TIMEELAPSED:	
                        cout<<"terminated: maximum time elapsed."<<endl;
                	break;
		case	ICP_SUCCESS:	
                	cout<<"succeeded: maximum deviation reached."<<endl;
                        break;
	}
	cout<<"elapsed time:\t"<<_dElapsedTime<<"ms"<<endl;
	cout<<"iterations:\t"<<unNoIter - 1<<endl;
	cout<<"deviation:\t"<<fDeviation<<endl;
}

double CIcpGpuCuda::getTime(void){
	return _dElapsedTime;
}

Matrix* CIcpGpuCuda::getMatrix(void){
	return final_matrix;
}

void CIcpGpuCuda::getCublasErr()
{
	hipblasStatus_t statusCUBLAS;
	statusCUBLAS = cublasGetError();
    if (statusCUBLAS != HIPBLAS_STATUS_SUCCESS) {
    	cout<<"CUBLAS error: ";
    	switch(statusCUBLAS){
    	case HIPBLAS_STATUS_NOT_INITIALIZED:	cout<<"CUBLAS library not initialized"<<endl;break;
    	case HIPBLAS_STATUS_ALLOC_FAILED:	cout<<"resource allocation failed"<<endl;break;
    	case HIPBLAS_STATUS_INVALID_VALUE:	cout<<"unsupported numerical value was passed to function"<<endl;break;
    	case HIPBLAS_STATUS_MAPPING_ERROR:	cout<<"access to GPU memory space failed"<<endl;break;
    	case HIPBLAS_STATUS_EXECUTION_FAILED:	cout<<"GPU program failed to execute"<<endl;break;
    	case HIPBLAS_STATUS_INTERNAL_ERROR:	cout<<"an internal CUBLAS operation failed"<<endl;break;
    	default:	cout<<"undefined error"<<endl;
    	}
    }
//    	cout<<"error number: "<<statusCUBLAS<<endl;
//        fprintf (stderr, "CUBLAS error.\n");};
}

void CIcpGpuCuda::getCudaErr(void){
	hipError_t error=hipGetLastError();
	cout<<hipGetErrorString(error)<<endl;
}

//kd-tree based nearest neighbor search, using a priority queue: no parameters are needed??
void CIcpGpuCuda::class_nns_priority(
		float* fDevScnX,						//scene point cloud
		float* fDevScnY,
		float* fDevScnZ,
		float* fDist,							//squared distance between pairs, for deviation calculation
		float* fDevSplit, 						//kd-tree: position of splitting plain (inner node)
		unsigned* unDevIdx, 					//kd-tree: index of point (leaf node)
		unsigned* unDevAxis, 					//kd-tree: axis where splitting plain locates (inner node)
		bool* bDevIsLeaf, 						//kd-tree: node type (both nodes)
		float* fDevLoBound,						//kd-tree: lower bounding box (inner node)
		float* fDevHiBound,						//kd-tree: higher bounding box (inner node)
//		unsigned* unDevResult,					//result, an array of scene point cloud indeces.
		unsigned* unMask,						//a 0-1 mask of pair and non-pairs.
		float* fDevMdlPairX,
		float* fDevMdlPairY,
		float* fDevMdlPairZ,
		float* fDevScnPairX,
		float* fDevScnPairY,
		float* fDevScnPairZ,
//		unsigned* temp,
		float fSearchRadius,
		unsigned unSize,
		unsigned unWidth,
		unsigned unQStep){						//for dubugging thread
	wrapper_nns_priority(/*fDevMdlX, fDevMdlY, fDevMdlZ,*/ fDevScnX, fDevScnY, fDevScnZ,
			fDist, fDevSplit, unDevIdx, unDevAxis, bDevIsLeaf, fDevLoBound, fDevHiBound,
			/*unDevResult,*/ unMask, fDevMdlPairX, fDevMdlPairY, fDevMdlPairZ, fDevScnPairX, fDevScnPairY, fDevScnPairZ,
			/*temp,*/ dimGrid, dimBlock, fSearchRadius, unSize, unWidth, unQStep);
                                }


//centralize a pointcloud
void CIcpGpuCuda::class_centralize(unsigned* unMask,
		float* fDevMdlPairX,
		float* fDevMdlPairY,
		float* fDevMdlPairZ,
		float* fDevScnPairX,
		float* fDevScnPairY,
		float* fDevScnPairZ,
		float fcm0,
		float fcm1,
		float fcm2,
		float fcs0,
		float fcs1,
		float fcs2,
		float* fCenteredModX,					//centered point cloud
		float* fCenteredModY,
		float* fCenteredModZ,
		float* fCenteredScnX,
		float* fCenteredScnY,
		float* fCenteredScnZ){
	wrapper_centralize(unMask,
			fDevMdlPairX,fDevMdlPairY,fDevMdlPairZ,fDevScnPairX,fDevScnPairY,fDevScnPairZ,
			fcm0,fcm1,fcm2,fcs0,fcs1,fcs2,
			fCenteredModX,fCenteredModY,fCenteredModZ,fCenteredScnX,fCenteredScnY,fCenteredScnZ,
			dimGrid, dimBlock);
}

//transform point cloud
void CIcpGpuCuda::class_transformation(float* fDevScnX,					//piont cloud to be transformed
		float* fDevScnY,
		float* fDevScnZ,
		float m00,	float m01,	float m02,	float m03,
		float m10,	float m11,	float m12,	float m13,
		float m20,	float m21,	float m22,	float m23){
	wrapper_transformation(fDevScnX, fDevScnY, fDevScnZ,
			m00,	m01,	m02,	m03,
			m10,	m11,	m12,	m13,
			m20,	m21,	m22,	m23,
			dimGrid, dimBlock);
}


void CIcpGpuCuda::setMinimums(float x, float y, float z){
    min_x = x;
    min_y = y;
    min_z = z;
}
        
Matrix** CIcpGpuCuda::getMatrices(){
    return matrices;
}

void CIcpGpuCuda::setTrans_Trans_inv(double tr[], double tr_inv[]){
    trans = new Matrix(4,4);
    trans_inv = new Matrix(4,4);
    (*trans)(1,1) = tr[0]; (*trans)(2,1)=tr[1];(*trans)(3,1)=tr[2];(*trans)(4,1)=tr[3];
    (*trans)(1,2) = tr[4]; (*trans)(2,2)=tr[5];(*trans)(3,2)=tr[6];(*trans)(4,2)=tr[7];
    (*trans)(1,3) = tr[8]; (*trans)(2,3)=tr[9];(*trans)(3,3)=tr[10];(*trans)(4,3)=tr[11];
    (*trans)(1,4) = tr[12]; (*trans)(2,4)=tr[13];(*trans)(3,4)=tr[14];(*trans)(4,4)=tr[15];

    (*trans_inv)(1,1) = tr_inv[0]; (*trans_inv)(2,1)=tr_inv[1];(*trans_inv)(3,1)=tr_inv[2];(*trans_inv)(4,1)=tr_inv[3];
    (*trans_inv)(1,2) = tr_inv[4]; (*trans_inv)(2,2)=tr_inv[5];(*trans_inv)(3,2)=tr_inv[6];(*trans_inv)(4,2)=tr_inv[7];
    (*trans_inv)(1,3) = tr_inv[8]; (*trans_inv)(2,3)=tr_inv[9];(*trans_inv)(3,3)=tr_inv[10];(*trans_inv)(4,3)=tr_inv[11];
    (*trans_inv)(1,4) = tr_inv[12]; (*trans_inv)(2,4)=tr_inv[13];(*trans_inv)(3,4)=tr_inv[14];(*trans_inv)(4,4)=tr_inv[15];

}


void CIcpGpuCuda::findNearestNeighbors(float fSearchRadius, unsigned unQStep){
                // We make a copy of the existing scene point cloud in order to transform it
                hipMemcpy(cngfDevScnX, fDevScnX, unSizeData*sizeof(float), hipMemcpyDeviceToDevice);
                hipMemcpy(cngfDevScnY, fDevScnY, unSizeData*sizeof(float), hipMemcpyDeviceToDevice);
                hipMemcpy(cngfDevScnZ, fDevScnZ, unSizeData*sizeof(float), hipMemcpyDeviceToDevice);
                //////

                class_transformation(cngfDevScnX, cngfDevScnY, cngfDevScnZ ,
                    (*trans_inv)(1,1), (*trans_inv)(1,2), (*trans_inv)(1,3), (*trans_inv)(1,4),
                    (*trans_inv)(2,1), (*trans_inv)(2,2), (*trans_inv)(2,3), (*trans_inv)(2,4),
                    (*trans_inv)(3,1), (*trans_inv)(3,2), (*trans_inv)(3,3), (*trans_inv)(3,4)
                    );

		class_nns_priority( cngfDevScnX, cngfDevScnY, cngfDevScnZ,
				fDist, fDevSplit, unDevIdx, unDevAxis, bDevIsLeaf, fDevLoBound, fDevHiBound,
				unMask, fDevMdlPairX, fDevMdlPairY, fDevMdlPairZ, fDevScnPairX, fDevScnPairY, fDevScnPairZ,
				fSearchRadius, unSizeData, _unWidth, unQStep);

                class_transformation(fDevMdlPairX, fDevMdlPairY, fDevMdlPairZ,
                    (*trans)(1,1), (*trans)(1,2), (*trans)(1,3), (*trans)(1,4),
                    (*trans)(2,1), (*trans)(2,2), (*trans)(2,3), (*trans)(2,4),
                    (*trans)(3,1), (*trans)(3,2), (*trans)(3,3), (*trans)(3,4)
                );

                class_transformation(fDevScnPairX, fDevScnPairY, fDevScnPairZ,
                    (*trans)(1,1), (*trans)(1,2), (*trans)(1,3), (*trans)(1,4),
                    (*trans)(2,1), (*trans)(2,2), (*trans)(2,3), (*trans)(2,4),
                    (*trans)(3,1), (*trans)(3,2), (*trans)(3,3), (*trans)(3,4)
                );


                /*fDevScnX
                cout<<"Trans Mat is : \n";
                printMatrix(trans);
                cout<<"Trans Inv Mat is : \n";
                printMatrix(trans_inv);
                */
                float tmpscn_x[10];
                float tmpscn_y[10];
                float tmpscn_z[10];
                float tmpmdl_x[10];
                float tmpmdl_y[10];
                float tmpmdl_z[10];
                hipMemcpy(tmpscn_x, fDevScnPairX, 10*sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(tmpscn_y, fDevScnPairY, 10*sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(tmpscn_z, fDevScnPairZ, 10*sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(tmpmdl_x, fDevMdlPairX, 10*sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(tmpmdl_y, fDevMdlPairY, 10*sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(tmpmdl_z, fDevMdlPairZ, 10*sizeof(float), hipMemcpyDeviceToHost);
/*                cout<<"------------------Point Pairs"<<endl;
                for(int i = 0 ; i < 10 ; ++i){
                    cout<<"Model "<<tmpmdl_x[i]<<" " << tmpmdl_y[i]<<" "<<tmpmdl_z[i];
                    cout<<" Scene "<<tmpscn_x[i]<<" " << tmpscn_y[i]<<" "<<tmpscn_z[i]<<endl;
                }
  */
}

Matrix CIcpGpuCuda::fillHomoMatrix(Matrix* R, double* dTranslation){
			//Fill result
    Matrix matrix(4,4);
    matrix(1,1) = (*R)(1,1);
    matrix(1,2) = (*R)(1,2);
    matrix(1,3) = (*R)(1,3);
    matrix(1,4) = dTranslation[0];

    matrix(2,1) = (*R)(2,1);
    matrix(2,2) = (*R)(2,2);
    matrix(2,3) = (*R)(2,3);
    matrix(2,4) = dTranslation[1];

    matrix(3,1) = (*R)(3,1);
    matrix(3,2) = (*R)(3,2);
    matrix(3,3) = (*R)(3,3);
    matrix(3,4) = dTranslation[2];

    matrix(4,1) = 0;
    matrix(4,2) = 0;
    matrix(4,3) = 0;
    matrix(4,4) = 1;
    return matrix;
}


void CIcpGpuCuda::computeCentroid(float* x, float* y, float* z, float*& center){
                     /*
                        hipblasSasum function works only with absolute values.
                        So I created an array of ones and used a dot product
                        in order to simulate a simple sum function
                    */

                    center[0] = hipblasSdot(unSizeData,x,1,ones,1);
                    center[0] /=unPairs;
                    center[1] = hipblasSdot(unSizeData,y,1,ones,1);
                    center[1] /=unPairs;
                    center[2] = hipblasSdot(unSizeData,z,1,ones,1);
                    center[2] /=unPairs;
}


Matrix CIcpGpuCuda::computeHMatrix(){
			Matrix H(3,3);
			H = 0.0;

			unsigned unSizeOfSec = 200000;//need to be tuned for best performance!				//+++ Fill by gpu +++
			if(unSizeData<=unSizeOfSec){
				H(1,1) = (double)hipblasSdot(unSizeData,fCenScnX,1,fCenModX,1);
				H(1,2) = (double)hipblasSdot(unSizeData,fCenScnX,1,fCenModY,1);
				H(1,3) = (double)hipblasSdot(unSizeData,fCenScnX,1,fCenModZ,1);
				H(2,1) = (double)hipblasSdot(unSizeData,fCenScnY,1,fCenModX,1);
				H(2,2) = (double)hipblasSdot(unSizeData,fCenScnY,1,fCenModY,1);
				H(2,3) = (double)hipblasSdot(unSizeData,fCenScnY,1,fCenModZ,1);
				H(3,1) = (double)hipblasSdot(unSizeData,fCenScnZ,1,fCenModX,1);
				H(3,2) = (double)hipblasSdot(unSizeData,fCenScnZ,1,fCenModY,1);
				H(3,3) = (double)hipblasSdot(unSizeData,fCenScnZ,1,fCenModZ,1);
			}
			else{
				unsigned unSections = (unsigned)(unSizeData/unSizeOfSec);//cout<<"unSections: "<<unSections<<endl;
				unsigned unStub = (unsigned)(unSizeData%unSizeOfSec);//cout<<"unStub: "<<unStub<<endl;
				for(unsigned i=0;i<unSections;i++){
					H(1,1) += (double)hipblasSdot(unSizeOfSec,fCenScnX+i*unSizeOfSec,1,fCenModX+i*unSizeOfSec,1);
					H(1,2) += (double)hipblasSdot(unSizeOfSec,fCenScnX+i*unSizeOfSec,1,fCenModY+i*unSizeOfSec,1);
					H(1,3) += (double)hipblasSdot(unSizeOfSec,fCenScnX+i*unSizeOfSec,1,fCenModZ+i*unSizeOfSec,1);
					H(2,1) += (double)hipblasSdot(unSizeOfSec,fCenScnY+i*unSizeOfSec,1,fCenModX+i*unSizeOfSec,1);
					H(2,2) += (double)hipblasSdot(unSizeOfSec,fCenScnY+i*unSizeOfSec,1,fCenModY+i*unSizeOfSec,1);
					H(2,3) += (double)hipblasSdot(unSizeOfSec,fCenScnY+i*unSizeOfSec,1,fCenModZ+i*unSizeOfSec,1);
					H(3,1) += (double)hipblasSdot(unSizeOfSec,fCenScnZ+i*unSizeOfSec,1,fCenModX+i*unSizeOfSec,1);
					H(3,2) += (double)hipblasSdot(unSizeOfSec,fCenScnZ+i*unSizeOfSec,1,fCenModY+i*unSizeOfSec,1);
					H(3,3) += (double)hipblasSdot(unSizeOfSec,fCenScnZ+i*unSizeOfSec,1,fCenModZ+i*unSizeOfSec,1);
				}
				if(unStub){
					H(1,1) += (double)hipblasSdot(unStub,fCenScnX+unSections*unSizeOfSec,1,fCenModX+unSections*unSizeOfSec,1);
					H(1,2) += (double)hipblasSdot(unStub,fCenScnX+unSections*unSizeOfSec,1,fCenModY+unSections*unSizeOfSec,1);
					H(1,3) += (double)hipblasSdot(unStub,fCenScnX+unSections*unSizeOfSec,1,fCenModZ+unSections*unSizeOfSec,1);
					H(2,1) += (double)hipblasSdot(unStub,fCenScnY+unSections*unSizeOfSec,1,fCenModX+unSections*unSizeOfSec,1);
					H(2,2) += (double)hipblasSdot(unStub,fCenScnY+unSections*unSizeOfSec,1,fCenModY+unSections*unSizeOfSec,1);
					H(2,3) += (double)hipblasSdot(unStub,fCenScnY+unSections*unSizeOfSec,1,fCenModZ+unSections*unSizeOfSec,1);
					H(3,1) += (double)hipblasSdot(unStub,fCenScnZ+unSections*unSizeOfSec,1,fCenModX+unSections*unSizeOfSec,1);
					H(3,2) += (double)hipblasSdot(unStub,fCenScnZ+unSections*unSizeOfSec,1,fCenModY+unSections*unSizeOfSec,1);
					H(3,3) += (double)hipblasSdot(unStub,fCenScnZ+unSections*unSizeOfSec,1,fCenModZ+unSections*unSizeOfSec,1);
				}
			}
			getCublasErr();
                        return H;

}

void CIcpGpuCuda::printMatrix(Matrix* mat){
    for(int i = 1 ; i < 5 ; ++i)
        cout<<(*mat)(i,1)<< " "<<(*mat)(i,2)<< " "<<(*mat)(i,3)<<
            " "<<(*mat)(i,4)<< endl;
}
