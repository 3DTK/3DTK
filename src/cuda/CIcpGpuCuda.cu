#include "hip/hip_runtime.h"
/** @file
 *  @brief GPU-ICP Algorithm
 *  @author Deyuan Qiu, University of Applied Sciences Bonn-Rhein-Sieg, Sankt Augustin, Germany.
 *                      Fraunhofer IAIS, Sankt Augustin, Germany.
 */


#include "CIcpGpuCuda_kernel.cuh"
#include "CIcpGpuCuda.cuh"
#include "CSystem.h"

void CIcpGpuCuda::init(unsigned unWidth, unsigned unHeight, unsigned max_iter)
{
	// Initialize CUTIL
	int d;
	hipSetDevice(0); // Since we have only one GPU I didn't initialize anything
	
	// It may cause problems if more GPUs take into account
	// The reason of doing that was initialization of it several times
	// Now it is not initialized several times and just once.

        unMaxIteration = max_iter;
        matrices = (Matrix**)malloc(sizeof(Matrix*) * max_iter);
        for(int i = 0 ; i < max_iter ; ++i){
            matrices[i] = new Matrix(4,4);
            Matrix* m = matrices[i];
            (*m)(1,1) = 1;(*m)(1,2) = 0;(*m)(1,3) = 0;(*m)(1,4) = 0;
            (*m)(2,1) = 0;(*m)(2,2) = 1;(*m)(2,3) = 0;(*m)(2,4) = 0;
            (*m)(3,1) = 0;(*m)(3,2) = 0;(*m)(3,3) = 1;(*m)(3,4) = 0;
            (*m)(4,1) = 0;(*m)(4,2) = 0;(*m)(4,3) = 0;(*m)(4,4) = 1;
        }

    // set data size
    setResolution(unWidth, unHeight);

    //    cout<<"unSizeData: "<<unSizeData<<endl;
    //    cout<<"Tree Size :"	<<TREESIZE<<endl;

    // Initialize CUBLAS
    hipblasStatus_t statusCUBLAS = cublasInit();
    if (statusCUBLAS != HIPBLAS_STATUS_SUCCESS) {
        cout<<"The error status is \n";
        cout<<statusCUBLAS<<endl;
        fprintf (stderr, "!!!! CUBLAS initialization error\n");
        exit(1);
    }

    // Initialize CUDPP
    CUDPPConfiguration config;
    config.datatype = CUDPP_FLOAT;
    config.algorithm = CUDPP_COMPACT;
    config.options = CUDPP_OPTION_FORWARD;
    result = cudppPlan(&compactplan, config, unSizeData, 1, 0);
    if (CUDPP_SUCCESS != result)	printf("Error creating CUDPPPlan\n");

    _unSizeTree = (unsigned)TREESIZE;
    CUDA_SAFE_CALL(hipHostMalloc((void**)&fSplit, _unSizeTree*sizeof(float)));
    CUDA_SAFE_CALL(hipHostMalloc((void**)&unIdx, _unSizeTree*sizeof(unsigned)));
    CUDA_SAFE_CALL(hipHostMalloc((void**)&unAxis, _unSizeTree*sizeof(unsigned)));
    CUDA_SAFE_CALL(hipHostMalloc((void**)&bIsLeaf, _unSizeTree*sizeof(bool)));
    CUDA_SAFE_CALL(hipHostMalloc((void**)&fLoBound, _unSizeTree*sizeof(float)));
    CUDA_SAFE_CALL(hipHostMalloc((void**)&fHiBound, _unSizeTree*sizeof(float)));

    CUDA_SAFE_CALL(hipHostMalloc((void**)&f4Mdl, unSizeData*sizeof(float4)));  // to be downloaded to texture

    // Host memory allocation
    CUDA_SAFE_CALL(hipHostMalloc((void**)&fHstScnX, unSizeData*sizeof(float)));
    CUDA_SAFE_CALL(hipHostMalloc((void**)&fHstScnY, unSizeData *sizeof(float)));
    CUDA_SAFE_CALL(hipHostMalloc((void**)&fHstScnZ, unSizeData*sizeof(float)));	//scene

    fHstScn[0]=fHstScnX;
    fHstScn[1]=fHstScnY;
    fHstScn[2]=fHstScnZ;

    CUDA_SAFE_CALL(hipHostMalloc((void**)&pNoPairs, sizeof(unsigned)));

    CSystem<double>::allocate(unSizeData, 3, h_idata);							//model

    // Device memory allocation
    CUDA_SAFE_CALL(hipMalloc((void**)&fDist, unSizeData*sizeof(float)));
    CUDA_SAFE_CALL(hipMalloc((void**)&fDistCpt, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&unMask, unSizeData*sizeof(unsigned)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fDevMdlPairX, unSizeData*sizeof(float)));	//pairs after shrinking
	CUDA_SAFE_CALL(hipMalloc((void**)&fDevMdlPairY, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fDevMdlPairZ, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fDevScnPairX, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fDevScnPairY, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fDevScnPairZ, unSizeData*sizeof(float)));
        /////////////// Added by Shams
     CUDA_SAFE_CALL(hipMalloc((void**)&cngfDevScnX,unSizeData*sizeof(float)));
     CUDA_SAFE_CALL(hipMalloc((void**)&cngfDevScnY,unSizeData*sizeof(float)));
     CUDA_SAFE_CALL(hipMalloc((void**)&cngfDevScnZ,unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&cngfDevMdlPairX, unSizeData*sizeof(float)));	//pairs after shrinking
	CUDA_SAFE_CALL(hipMalloc((void**)&cngfDevMdlPairY, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&cngfDevMdlPairZ, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&cngfDevScnPairX, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&cngfDevScnPairY, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&cngfDevScnPairZ, unSizeData*sizeof(float)));

        ///////////////
	CUDA_SAFE_CALL(hipMalloc((void**)&fCenModX, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fCenModY, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fCenModZ, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fCenScnX, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fCenScnY, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fCenScnZ, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&unNoPairs, sizeof(unsigned)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fDevScnX, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fDevScnY, unSizeData*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fDevScnZ, unSizeData*sizeof(float)));

	CUDA_SAFE_CALL(hipMalloc((void**)&fDevSplit, _unSizeTree*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&unDevIdx, _unSizeTree*sizeof(unsigned)));
	CUDA_SAFE_CALL(hipMalloc((void**)&unDevAxis, _unSizeTree*sizeof(unsigned)));
	CUDA_SAFE_CALL(hipMalloc((void**)&bDevIsLeaf, _unSizeTree*sizeof(bool)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fDevLoBound, _unSizeTree*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fDevHiBound, _unSizeTree*sizeof(float)));

	CUDA_SAFE_CALL(hipMallocArray(&cuArray, &cuDesc, _unWidth, _unHeight));		//to be bound to texture

	// Initialize states
	fMaxProcTime		=	0.0f;
	fMaxDeviation		=	0.0f;
	_fSearchRadiusMax	=	0.0f;
	_fSearchRadiusMin	=	0.0f;
	_fRadiusStep		=	0.0f;
	_unNoQSizeStep		=	0;
	_dElapsedTime		=	0.0;

        /*
	    * Array of ones to be used instead of abs sum
        */
	hipHostMalloc((void**)&temp_ones, unSizeData*sizeof(float));
	hipMalloc((void**)&ones, unSizeData*sizeof(float));	// Array of ones
     for(int i = 0; i < unSizeData ; ++i)temp_ones[i] = 1.0f;
	hipMemcpy(ones, temp_ones, unSizeData*sizeof(float), hipMemcpyHostToDevice);
}

CIcpGpuCuda::~CIcpGpuCuda(){
    /////////////
	// tidy up
    /////////////
	CUDA_SAFE_CALL(hipUnbindTexture(refTex));
	CUDA_SAFE_CALL(hipFreeArray(cuArray));
	CUDA_SAFE_CALL(hipFree(fDevSplit));
	CUDA_SAFE_CALL(hipFree(unDevIdx));
	CUDA_SAFE_CALL(hipFree(unDevAxis));
	CUDA_SAFE_CALL(hipFree(bDevIsLeaf));
	CUDA_SAFE_CALL(hipFree(fDevLoBound));
	CUDA_SAFE_CALL(hipFree(fDevHiBound));
	CUDA_SAFE_CALL(hipFree(fDevScnX));
	CUDA_SAFE_CALL(hipFree(fDevScnY));
	CUDA_SAFE_CALL(hipFree(fDevScnZ));
	CUDA_SAFE_CALL(hipFree(fDist));
	CUDA_SAFE_CALL(hipFree(fDistCpt));
	CUDA_SAFE_CALL(hipFree(fCenModX));
	CUDA_SAFE_CALL(hipFree(fCenModY));
	CUDA_SAFE_CALL(hipFree(fCenModZ));
	CUDA_SAFE_CALL(hipFree(fCenScnX));
	CUDA_SAFE_CALL(hipFree(fCenScnY));
	CUDA_SAFE_CALL(hipFree(fCenScnZ));
	CUDA_SAFE_CALL(hipFree(unMask));
	CUDA_SAFE_CALL(hipFree(fDevMdlPairX));
	CUDA_SAFE_CALL(hipFree(fDevMdlPairY));
	CUDA_SAFE_CALL(hipFree(fDevMdlPairZ));
	CUDA_SAFE_CALL(hipFree(fDevScnPairX));
	CUDA_SAFE_CALL(hipFree(fDevScnPairY));
	CUDA_SAFE_CALL(hipFree(fDevScnPairZ));
     CUDA_SAFE_CALL(hipFree(cngfDevScnX));
     CUDA_SAFE_CALL(hipFree(cngfDevScnY));
     CUDA_SAFE_CALL(hipFree(cngfDevScnZ));
	CUDA_SAFE_CALL(hipFree(cngfDevMdlPairX));
	CUDA_SAFE_CALL(hipFree(cngfDevMdlPairY));
	CUDA_SAFE_CALL(hipFree(cngfDevMdlPairZ));
	CUDA_SAFE_CALL(hipFree(cngfDevScnPairX));
	CUDA_SAFE_CALL(hipFree(cngfDevScnPairY));
	CUDA_SAFE_CALL(hipFree(cngfDevScnPairZ));

	CUDA_SAFE_CALL(hipFree(unNoPairs));
	CUDA_SAFE_CALL(hipFree(ones));


	CUDA_SAFE_CALL(hipHostFree(fSplit));
	CUDA_SAFE_CALL(hipHostFree(unIdx));
	CUDA_SAFE_CALL(hipHostFree(unAxis));
	CUDA_SAFE_CALL(hipHostFree(bIsLeaf));
	CUDA_SAFE_CALL(hipHostFree(fHstScnX));
	CUDA_SAFE_CALL(hipHostFree(fHstScnY));
	CUDA_SAFE_CALL(hipHostFree(fHstScnZ));
	CUDA_SAFE_CALL(hipHostFree(fLoBound));
	CUDA_SAFE_CALL(hipHostFree(fHiBound));
	CUDA_SAFE_CALL(hipHostFree(pNoPairs));
	CUDA_SAFE_CALL(hipHostFree(f4Mdl));
	CUDA_SAFE_CALL(hipHostFree(temp_ones));

	free(h_idata);


    /////////
    // Exit
    /////////
    bool bShutDownSuccess = true;

    // Done with CUDPP
    result = cudppDestroyPlan(compactplan);
    if (CUDPP_SUCCESS != result){
    	printf("Error destroying CUDPPPlan\n");
    	bShutDownSuccess = false;
    }

    // Done with CUBLAS
    hipblasStatus_t statusCUBLAS = cublasShutdown();
    if (statusCUBLAS != HIPBLAS_STATUS_SUCCESS) {
        fprintf (stderr, "!!!! shutdown error (A)\n");
        bShutDownSuccess = false;
    }

    // Done with ANN
    annClose();

}

void CIcpGpuCuda::setResolution(unsigned unWidth, unsigned unHeight){
	if (unWidth>0 && unHeight>0) {
		_unWidth = unWidth;
		_unHeight = unHeight;
		unSizeData = _unWidth*_unHeight;

		//configure block and grid size
		unsigned unBlockSize = (unsigned)BLOCKSIZE;	//192
		if(unSizeData<=64){
			unNoThreads = 64;
			unNoBlocks = 1;
		}
		else if(unSizeData<=128){
			unNoThreads = 128;
			unNoBlocks = 1;
		}
		else if(unSizeData<=unBlockSize){
			unNoThreads = unBlockSize;
			unNoBlocks = 1;
		}
		else{
			unNoThreads = unBlockSize;
			if(unSizeData%unBlockSize)	unNoBlocks = unSizeData/unBlockSize + 1;
			else	unNoBlocks = unSizeData/unBlockSize;
		}
		cout<<"unNoThreads: "<<unNoThreads<<'\t'<<"unNoBlocks: "<<unNoBlocks<<endl;
		dimBlock.x=unNoThreads;
		dimBlock.y=1;
		dimBlock.z=1;
		dimGrid.x=unNoBlocks;
		dimGrid.y=1;
		dimGrid.z=1;
	}
	else{
		cout<<"Each aspect of resolution must be bigger than 0."<<endl;
		exit(1);
	}
}

void CIcpGpuCuda::setMaxIteration(unsigned unTimes){
	if(unTimes>0)	unMaxIteration = unTimes;
	else{
		cout<<"Error setting maximum iterations."<<endl;
		exit(1);
	}
}

void CIcpGpuCuda::setMaxProcTime(double dMilliseconds){
	if(dMilliseconds>0)	fMaxProcTime = dMilliseconds;
	else{
		cout<<"Error setting maximum processing time."<<endl;
		exit(1);
	}
}

void CIcpGpuCuda::setMaxDeviation(double fDeviation){
	if(fDeviation>0)	fMaxDeviation = fDeviation;
	else{
		cout<<"Error setting maximum deviation."<<endl;
		exit(1);
	}
}

void CIcpGpuCuda::setSearchRadius(float fRadiusMax, float fRadiusMin, unsigned unIterations){
	if((fRadiusMax>=fRadiusMin)&&(unIterations>0)){
		_fSearchRadiusMax = fRadiusMax;
		_fSearchRadiusMin = fRadiusMin;
		_fRadiusStep = (_fSearchRadiusMax-_fSearchRadiusMin)/(float)unIterations;
		_unIterations = unIterations;
		_unNoQSizeStep = unIterations/(unsigned)NO_QSIZE;
	}
	else{
		cout<<"Error setting search radius."<<endl;
		exit(1);
	}
}

float** CIcpGpuCuda::getScenePointer() {
	return fHstScn;
}

double** CIcpGpuCuda::getModelPointer(void){
	return h_idata;
}

unsigned CIcpGpuCuda::getSize(void){
	return unSizeData;
}

void CIcpGpuCuda::setTreePointer(ANNkd_tree *tree){
    kdTree = tree;
}
void CIcpGpuCuda::getTreePointer(ANNkd_tree *&tree ){
    tree = kdTree;
}


void CIcpGpuCuda::setTree()
{

	// preparation
	st = new ANNkdStats();
	kdTree->getStats(*st);
	int nDepth = st->depth;
     // cout<<"level of the tree: "<<st->depth<<" (counted from 0)"<<endl;
	// decide size of array to be uploaded to GPU
	unSizeTree = depth2size(nDepth);
	// cout<<"unSizeTree: "<<unSizeTree<<endl;

	if (unSizeTree>_unSizeTree) {
		cout << "Not enough memory for tree construction. Tree size must be smaller than "
		     << unSizeTree <<endl;
		exit(1);
	}

	// rearrange
	ANNkd_split* pRoot = (ANNkd_split*)kdTree->getRoot();
	if (unSizeData>1) {
	  rearrange(pRoot, 1);
	} else {
	  cout<<"Not enough points in the tree."<<endl;
	  exit(1);
	}

	// download the tree
	CUDA_SAFE_CALL(hipMemcpy(fDevSplit, fSplit, unSizeTree*sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(unDevIdx, unIdx, unSizeTree*sizeof(unsigned), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(unDevAxis, unAxis, unSizeTree*sizeof(unsigned), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(bDevIsLeaf, bIsLeaf, unSizeTree*sizeof(bool), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(fDevLoBound, fLoBound, unSizeTree*sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(fDevHiBound, fHiBound, unSizeTree*sizeof(float), hipMemcpyHostToDevice));

	// clean up
//	delete kdTree;
	delete st;
}

void CIcpGpuCuda::setModel()
{
	// using texture memory
	for(unsigned i = 0; i < unSizeData; i++) {	// type cast
	  f4Mdl[i].x=(float)h_idata[i][0];
	  f4Mdl[i].y=(float)h_idata[i][1];
	  f4Mdl[i].z=(float)h_idata[i][2];
	}
	hipMemcpyToArray(cuArray,0,0,f4Mdl,unSizeData*sizeof(float4),hipMemcpyHostToDevice);
	hipBindTextureToArray(refTex,cuArray);
}

void CIcpGpuCuda::setScene()
{
	CUDA_SAFE_CALL(hipMemcpy(fDevScnX, fHstScnX, unSizeData*sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(fDevScnY, fHstScnY, unSizeData*sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(fDevScnZ, fHstScnZ, unSizeData*sizeof(float), hipMemcpyHostToDevice));
}

inline int CIcpGpuCuda::depth2size(int nDepth)
{
	double nSize = 0;
	for(int i = 0;i <= nDepth; i++)
	{
		nSize += pow(2.0,(double)i);
	}
	return (int)(nSize+0.5);
}

void CIcpGpuCuda::rearrange(ANNkd_ptr root, unsigned unStart)
{
	if(root!=NULL && root!=KD_TRIVIAL){
		if (root->isLeaf()) {
			bIsLeaf[unStart-1] = true;
			unIdx[unStart-1] = (unsigned)(((ANNkd_leaf*)root)->getIdxArray())[0];
		}
		else {
			ANNkd_ptr nL = ((ANNkd_split*)root)->getLeftChild();
			ANNkd_ptr nR = ((ANNkd_split*)root)->getRightChild();

			bIsLeaf[unStart-1] = false;
			fSplit[unStart-1] = (float)((ANNkd_split*)root)->getCutVal();
			unAxis[unStart-1] = (unsigned)((ANNkd_split*)root)->getCutDim();
			fLoBound[unStart-1] = ((ANNkd_split*)root)->getLoBound();
			fHiBound[unStart-1] = ((ANNkd_split*)root)->getHiBound();
			rearrange(nL, unStart*2);
			rearrange(nR, unStart*2+1);
		}
	}
}

void CIcpGpuCuda::setPointClouds(void){
	setTree();
	setModel();
	setScene();
}

void CIcpGpuCuda::iteration(){

	//////////////////////////
	// prepare for iteration
	//////////////////////////

	EnumIcpState icpStat = ICP_PROCESSING;

	float fSearchRadius = _fSearchRadiusMax;
	unsigned unNoIter = 0;
	float fDeviation = 0.0;
	unsigned unQStep = 0;

	final_matrix = new Matrix(4,4);
	(*final_matrix)(1,1) = 1.0;(*final_matrix)(1,2) = 0.0;(*final_matrix)(1,3) = 0.0;(*final_matrix)(1,4) = 0.0;
	(*final_matrix)(2,1) = 0.0;(*final_matrix)(2,2) = 1.0;(*final_matrix)(2,3) = 0.0;(*final_matrix)(2,4) = 0.0;
	(*final_matrix)(3,1) = 0.0;(*final_matrix)(3,2) = 0.0;(*final_matrix)(3,3) = 1.0;(*final_matrix)(3,4) = 0.0;
	(*final_matrix)(4,1) = 0.0;(*final_matrix)(4,2) = 0.0;(*final_matrix)(4,3) = 0.0;(*final_matrix)(4,4) = 1.0;

	Matrix matrix(4,4);
	
	init_time = clock();

     // The main loop of ICP
	while(icpStat == ICP_PROCESSING){
       if (unNoIter <= _unIterations)
		unQStep = unNoIter/_unNoQSizeStep;

            findNearestNeighbors(fSearchRadius, unQStep);

            result=cudppCompact(compactplan, fDistCpt, (size_t*)unNoPairs,
                     fDist, unMask, (size_t)unSizeData);

	    if (CUDPP_SUCCESS != result)
                printf("Error cudppCompact\n");
            CUDA_SAFE_CALL(hipMemcpy(pNoPairs, unNoPairs, sizeof(unsigned), hipMemcpyDeviceToHost));
            unPairs = *pNoPairs;

            if (unPairs) {
		//////////////////////
		// transform estimation
		//////////////////////

        	// Compute centroids (assume all data are non-negative)
		float *fCm = new float(3);
		float *fCs = new float(3);


                computeCentroid(fDevMdlPairX,fDevMdlPairY,fDevMdlPairZ, fCm);
                computeCentroid(fDevScnPairX,fDevScnPairY,fDevScnPairZ, fCs);

                fDeviation = hipblasSdot(unSizeData,fDistCpt,1,ones,1);
                fDeviation /= unPairs;
                getCublasErr();

		// check for termination conditions
		unNoIter++;
		if(unNoIter<_unIterations)	
			fSearchRadius-=_fRadiusStep;
		if(fDeviation <= fMaxDeviation)
			icpStat = ICP_SUCCESS;
		else if(unNoIter >= unMaxIteration + 1) // unNoIter starts from 1
			icpStat = ICP_MAXITERATIONS;
		else if ( (double)(clock() - init_time)/ (double)CLOCKS_PER_SEC * 1000>= fMaxProcTime )
			icpStat = ICP_TIMEELAPSED;
		else
		{
			//Calculate centered point pairs
			class_centralize(unMask,
					fDevMdlPairX,fDevMdlPairY,fDevMdlPairZ,fDevScnPairX,fDevScnPairY,fDevScnPairZ,
					fCm[0],fCm[1],fCm[2],fCs[0],fCs[1],fCs[2],
					fCenModX,fCenModY,fCenModZ,fCenScnX,fCenScnY,fCenScnZ);

			//Fill H matrix
                        Matrix H(3,3);
                        H = computeHMatrix();

			//SVD
			Matrix U(3,3);
			DiagonalMatrix Lamda(3);
			Matrix V(3,3);
			SVD(H,Lamda,U,V);

			//Get rotation
			Matrix R(3,3);
			R = V*(U.t());

			// Calculate translation
			double dTranslation[3];
			ColumnVector col_vec(3);
			for(unsigned j = 0; j < 3; j++)
				col_vec(j+1) = fCs[j];

			ColumnVector r_time_colVec = ColumnVector(R*col_vec);

			dTranslation[0] = fCm[0] - r_time_colVec(1);
			dTranslation[1] = fCm[1] - r_time_colVec(2);
			dTranslation[2] = fCm[2] - r_time_colVec(3);

               matrix = fillHomoMatrix(&R,dTranslation);

			*final_matrix = matrix * (*final_matrix);

                        for(int i = 1 ; i < 5 ; ++i)
                            for(int j = 1; j < 5 ; ++j){
                                (*matrices[unNoIter - 1])(i,j) = (matrix)(i,j);
                            }

			/////////////
			// transform
			/////////////

			class_transformation(fDevScnX, fDevScnY, fDevScnZ,
					(float)matrix(1,1),	(float)matrix(1,2),	(float)matrix(1,3),	(float)matrix(1,4),
					(float)matrix(2,1),	(float)matrix(2,2),	(float)matrix(2,3),	(float)matrix(2,4),
					(float)matrix(3,1),	(float)matrix(3,2),	(float)matrix(3,3),	(float)matrix(3,4));


			///////////////
			// termination
			///////////////

			}
		}//if(unPairs)
		else	icpStat = ICP_NOTMATCHABLE;
	}//while(icpStat == ICP_PROCESSING)

	_dElapsedTime = (double)(clock() - init_time)/(double)CLOCKS_PER_SEC * 1000.0;		//temporary
	cout<<"=========="<<endl;
	switch(icpStat)
	{
		case	ICP_LIMIT:	
        		cout<<"terminated: convergent limit reached."<<endl;
                        break;
		case	ICP_NOTMATCHABLE:	
                        cout<<"terminated: point clounds not matchable."<<endl;
				    /*
    	(*final_matrix)(1,1) = 1.0;(*final_matrix)(1,2) = 0.0;(*final_matrix)(1,3) = 0.0;(*final_matrix)(1,4) = 0.0;
	(*final_matrix)(2,1) = 0.0;(*final_matrix)(2,2) = 1.0;(*final_matrix)(2,3) = 0.0;(*final_matrix)(2,4) = 0.0;
	(*final_matrix)(3,1) = 0.0;(*final_matrix)(3,2) = 0.0;(*final_matrix)(3,3) = 1.0;(*final_matrix)(3,4) = 0.0;
	(*final_matrix)(4,1) = 0.0;(*final_matrix)(4,2) = 0.0;(*final_matrix)(4,3) = 0.0;(*final_matrix)(4,4) = 1.0;
*/
                        break;
		case	ICP_MAXITERATIONS:	
                        cout<<"terminated: maximum iteration exceeds."<<endl;
                        break;
		case	ICP_TIMEELAPSED:	
                        cout<<"terminated: maximum time elapsed."<<endl;
                	break;
		case	ICP_SUCCESS:	
                	cout<<"succeeded: maximum deviation reached."<<endl;
                        break;
	}
	cout<<"elapsed time:\t"<<_dElapsedTime<<"ms"<<endl;
	cout<<"iterations:\t"<<unNoIter - 1<<endl;
	cout<<"deviation:\t"<<fDeviation<<endl;
}

double CIcpGpuCuda::getTime(void){
	return _dElapsedTime;
}

Matrix* CIcpGpuCuda::getMatrix(void){
	return final_matrix;
}

void CIcpGpuCuda::getCublasErr()
{
	hipblasStatus_t statusCUBLAS;
	statusCUBLAS = cublasGetError();
    if (statusCUBLAS != HIPBLAS_STATUS_SUCCESS) {
    	cout<<"CUBLAS error: ";
    	switch(statusCUBLAS){
    	case HIPBLAS_STATUS_NOT_INITIALIZED:	cout<<"CUBLAS library not initialized"<<endl;break;
    	case HIPBLAS_STATUS_ALLOC_FAILED:	cout<<"resource allocation failed"<<endl;break;
    	case HIPBLAS_STATUS_INVALID_VALUE:	cout<<"unsupported numerical value was passed to function"<<endl;break;
    	case HIPBLAS_STATUS_MAPPING_ERROR:	cout<<"access to GPU memory space failed"<<endl;break;
    	case HIPBLAS_STATUS_EXECUTION_FAILED:	cout<<"GPU program failed to execute"<<endl;break;
    	case HIPBLAS_STATUS_INTERNAL_ERROR:	cout<<"an internal CUBLAS operation failed"<<endl;break;
    	default:	cout<<"undefined error"<<endl;
    	}
    }
//    	cout<<"error number: "<<statusCUBLAS<<endl;
//        fprintf (stderr, "CUBLAS error.\n");};
}

void CIcpGpuCuda::getCudaErr(void){
	hipError_t error=hipGetLastError();
	cout<<hipGetErrorString(error)<<endl;
}

//kd-tree based nearest neighbor search, using a priority queue: no parameters are needed??
void CIcpGpuCuda::class_nns_priority(
		float* fDevScnX,						//scene point cloud
		float* fDevScnY,
		float* fDevScnZ,
		float* fDist,							//squared distance between pairs, for deviation calculation
		float* fDevSplit, 						//kd-tree: position of splitting plain (inner node)
		unsigned* unDevIdx, 					//kd-tree: index of point (leaf node)
		unsigned* unDevAxis, 					//kd-tree: axis where splitting plain locates (inner node)
		bool* bDevIsLeaf, 						//kd-tree: node type (both nodes)
		float* fDevLoBound,						//kd-tree: lower bounding box (inner node)
		float* fDevHiBound,						//kd-tree: higher bounding box (inner node)
		unsigned* unMask,						//a 0-1 mask of pair and non-pairs.
		float* fDevMdlPairX,
		float* fDevMdlPairY,
		float* fDevMdlPairZ,
		float* fDevScnPairX,
		float* fDevScnPairY,
		float* fDevScnPairZ,
		float fSearchRadius,
		unsigned unSize,
		unsigned unWidth,
		unsigned unQStep)
{
   wrapper_nns_priority( fDevScnX, fDevScnY, fDevScnZ,
		             	fDist, fDevSplit, unDevIdx, unDevAxis, bDevIsLeaf, fDevLoBound, fDevHiBound,
			          unMask,
					fDevMdlPairX, fDevMdlPairY, fDevMdlPairZ, fDevScnPairX, fDevScnPairY, fDevScnPairZ,
			          dimGrid, dimBlock, fSearchRadius,
					unSize, unWidth,
					unQStep );
}


//centralize a pointcloud
void CIcpGpuCuda::class_centralize(unsigned* unMask,
		float* fDevMdlPairX,
		float* fDevMdlPairY,
		float* fDevMdlPairZ,
		float* fDevScnPairX,
		float* fDevScnPairY,
		float* fDevScnPairZ,
		float fcm0,
		float fcm1,
		float fcm2,
		float fcs0,
		float fcs1,
		float fcs2,
		float* fCenteredModX,					//centered point cloud
		float* fCenteredModY,
		float* fCenteredModZ,
		float* fCenteredScnX,
		float* fCenteredScnY,
		float* fCenteredScnZ){
	wrapper_centralize(unMask,
			fDevMdlPairX,fDevMdlPairY,fDevMdlPairZ,fDevScnPairX,fDevScnPairY,fDevScnPairZ,
			fcm0,fcm1,fcm2,fcs0,fcs1,fcs2,
			fCenteredModX,fCenteredModY,fCenteredModZ,fCenteredScnX,fCenteredScnY,fCenteredScnZ,
			dimGrid, dimBlock);
}

// transform point cloud
void CIcpGpuCuda::class_transformation(float* fDevScnX,	// point cloud to be transformed
		                             float* fDevScnY,
		                             float* fDevScnZ,
		                             float m00, float m01, float m02, float m03,
		                             float m10, float m11, float m12, float m13,
		                             float m20, float m21, float m22, float m23)
{
   wrapper_transformation(fDevScnX, fDevScnY, fDevScnZ,
			           m00, m01, m02, m03,
			           m10, m11, m12, m13,
                          m20, m21, m22, m23,
                          dimGrid, dimBlock);
}


void CIcpGpuCuda::setMinimums(float x, float y, float z)
{
    min_x = x;
    min_y = y;
    min_z = z;
}
        
Matrix** CIcpGpuCuda::getMatrices(){
    return matrices;
}

void CIcpGpuCuda::setTrans_Trans_inv(const double tr[], const double tr_inv[]){
    trans = new Matrix(4,4);
    trans_inv = new Matrix(4,4);
    (*trans)(1,1) = tr[0]; (*trans)(2,1)=tr[1];(*trans)(3,1)=tr[2];(*trans)(4,1)=tr[3];
    (*trans)(1,2) = tr[4]; (*trans)(2,2)=tr[5];(*trans)(3,2)=tr[6];(*trans)(4,2)=tr[7];
    (*trans)(1,3) = tr[8]; (*trans)(2,3)=tr[9];(*trans)(3,3)=tr[10];(*trans)(4,3)=tr[11];
    (*trans)(1,4) = tr[12]; (*trans)(2,4)=tr[13];(*trans)(3,4)=tr[14];(*trans)(4,4)=tr[15];

    (*trans_inv)(1,1) = tr_inv[0]; (*trans_inv)(2,1)=tr_inv[1];(*trans_inv)(3,1)=tr_inv[2];(*trans_inv)(4,1)=tr_inv[3];
    (*trans_inv)(1,2) = tr_inv[4]; (*trans_inv)(2,2)=tr_inv[5];(*trans_inv)(3,2)=tr_inv[6];(*trans_inv)(4,2)=tr_inv[7];
    (*trans_inv)(1,3) = tr_inv[8]; (*trans_inv)(2,3)=tr_inv[9];(*trans_inv)(3,3)=tr_inv[10];(*trans_inv)(4,3)=tr_inv[11];
    (*trans_inv)(1,4) = tr_inv[12]; (*trans_inv)(2,4)=tr_inv[13];(*trans_inv)(3,4)=tr_inv[14];(*trans_inv)(4,4)=tr_inv[15];

}


void CIcpGpuCuda::findNearestNeighbors(float fSearchRadius, unsigned unQStep)
{
                // We make a copy of the existing scene point cloud in order to transform it
                hipMemcpy(cngfDevScnX, fDevScnX, unSizeData*sizeof(float), hipMemcpyDeviceToDevice);
                hipMemcpy(cngfDevScnY, fDevScnY, unSizeData*sizeof(float), hipMemcpyDeviceToDevice);
                hipMemcpy(cngfDevScnZ, fDevScnZ, unSizeData*sizeof(float), hipMemcpyDeviceToDevice);
                //////

                class_transformation(cngfDevScnX, cngfDevScnY, cngfDevScnZ ,
                    (*trans_inv)(1,1), (*trans_inv)(1,2), (*trans_inv)(1,3), (*trans_inv)(1,4),
                    (*trans_inv)(2,1), (*trans_inv)(2,2), (*trans_inv)(2,3), (*trans_inv)(2,4),
                    (*trans_inv)(3,1), (*trans_inv)(3,2), (*trans_inv)(3,3), (*trans_inv)(3,4)
                    );

		class_nns_priority( cngfDevScnX, cngfDevScnY, cngfDevScnZ,
				fDist, fDevSplit, unDevIdx, unDevAxis, bDevIsLeaf, fDevLoBound, fDevHiBound,
				unMask, fDevMdlPairX, fDevMdlPairY, fDevMdlPairZ, fDevScnPairX, fDevScnPairY, fDevScnPairZ,
				fSearchRadius, unSizeData, _unWidth, unQStep);

                class_transformation(fDevMdlPairX, fDevMdlPairY, fDevMdlPairZ,
                    (*trans)(1,1), (*trans)(1,2), (*trans)(1,3), (*trans)(1,4),
                    (*trans)(2,1), (*trans)(2,2), (*trans)(2,3), (*trans)(2,4),
                    (*trans)(3,1), (*trans)(3,2), (*trans)(3,3), (*trans)(3,4)
                );

                class_transformation(fDevScnPairX, fDevScnPairY, fDevScnPairZ,
                    (*trans)(1,1), (*trans)(1,2), (*trans)(1,3), (*trans)(1,4),
                    (*trans)(2,1), (*trans)(2,2), (*trans)(2,3), (*trans)(2,4),
                    (*trans)(3,1), (*trans)(3,2), (*trans)(3,3), (*trans)(3,4)
                );


                /*fDevScnX
                cout<<"Trans Mat is : \n";
                printMatrix(trans);
                cout<<"Trans Inv Mat is : \n";
                printMatrix(trans_inv);
                */
                float tmpscn_x[10];
                float tmpscn_y[10];
                float tmpscn_z[10];
                float tmpmdl_x[10];
                float tmpmdl_y[10];
                float tmpmdl_z[10];
                hipMemcpy(tmpscn_x, fDevScnPairX, 10*sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(tmpscn_y, fDevScnPairY, 10*sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(tmpscn_z, fDevScnPairZ, 10*sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(tmpmdl_x, fDevMdlPairX, 10*sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(tmpmdl_y, fDevMdlPairY, 10*sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(tmpmdl_z, fDevMdlPairZ, 10*sizeof(float), hipMemcpyDeviceToHost);
}

Matrix CIcpGpuCuda::fillHomoMatrix(Matrix* R, double* dTranslation){
			//Fill result
    Matrix matrix(4,4);
    matrix(1,1) = (*R)(1,1);
    matrix(1,2) = (*R)(1,2);
    matrix(1,3) = (*R)(1,3);
    matrix(1,4) = dTranslation[0];

    matrix(2,1) = (*R)(2,1);
    matrix(2,2) = (*R)(2,2);
    matrix(2,3) = (*R)(2,3);
    matrix(2,4) = dTranslation[1];

    matrix(3,1) = (*R)(3,1);
    matrix(3,2) = (*R)(3,2);
    matrix(3,3) = (*R)(3,3);
    matrix(3,4) = dTranslation[2];

    matrix(4,1) = 0;
    matrix(4,2) = 0;
    matrix(4,3) = 0;
    matrix(4,4) = 1;
    return matrix;
}


void CIcpGpuCuda::computeCentroid(float* x, float* y, float* z, float*& center){
                     /*
                        hipblasSasum function works only with absolute values.
                        So I created an array of ones and used a dot product
                        in order to simulate a simple sum function
                    */

                    center[0] = hipblasSdot(unSizeData,x,1,ones,1);
                    center[0] /=unPairs;
                    center[1] = hipblasSdot(unSizeData,y,1,ones,1);
                    center[1] /=unPairs;
                    center[2] = hipblasSdot(unSizeData,z,1,ones,1);
                    center[2] /=unPairs;
}


Matrix CIcpGpuCuda::computeHMatrix(){
			Matrix H(3,3);
			H = 0.0;

			unsigned unSizeOfSec = 200000; // need to be tuned for best performance!// +++ Fill by gpu +++
			if(unSizeData<=unSizeOfSec){
				H(1,1) = (double)hipblasSdot(unSizeData,fCenScnX,1,fCenModX,1);
				H(1,2) = (double)hipblasSdot(unSizeData,fCenScnX,1,fCenModY,1);
				H(1,3) = (double)hipblasSdot(unSizeData,fCenScnX,1,fCenModZ,1);
				H(2,1) = (double)hipblasSdot(unSizeData,fCenScnY,1,fCenModX,1);
				H(2,2) = (double)hipblasSdot(unSizeData,fCenScnY,1,fCenModY,1);
				H(2,3) = (double)hipblasSdot(unSizeData,fCenScnY,1,fCenModZ,1);
				H(3,1) = (double)hipblasSdot(unSizeData,fCenScnZ,1,fCenModX,1);
				H(3,2) = (double)hipblasSdot(unSizeData,fCenScnZ,1,fCenModY,1);
				H(3,3) = (double)hipblasSdot(unSizeData,fCenScnZ,1,fCenModZ,1);
			}
			else{
				unsigned unSections = (unsigned)(unSizeData/unSizeOfSec);//cout<<"unSections: "<<unSections<<endl;
				unsigned unStub = (unsigned)(unSizeData%unSizeOfSec);//cout<<"unStub: "<<unStub<<endl;
				for(unsigned i=0;i<unSections;i++){
					H(1,1) += (double)hipblasSdot(unSizeOfSec,fCenScnX+i*unSizeOfSec,1,fCenModX+i*unSizeOfSec,1);
					H(1,2) += (double)hipblasSdot(unSizeOfSec,fCenScnX+i*unSizeOfSec,1,fCenModY+i*unSizeOfSec,1);
					H(1,3) += (double)hipblasSdot(unSizeOfSec,fCenScnX+i*unSizeOfSec,1,fCenModZ+i*unSizeOfSec,1);
					H(2,1) += (double)hipblasSdot(unSizeOfSec,fCenScnY+i*unSizeOfSec,1,fCenModX+i*unSizeOfSec,1);
					H(2,2) += (double)hipblasSdot(unSizeOfSec,fCenScnY+i*unSizeOfSec,1,fCenModY+i*unSizeOfSec,1);
					H(2,3) += (double)hipblasSdot(unSizeOfSec,fCenScnY+i*unSizeOfSec,1,fCenModZ+i*unSizeOfSec,1);
					H(3,1) += (double)hipblasSdot(unSizeOfSec,fCenScnZ+i*unSizeOfSec,1,fCenModX+i*unSizeOfSec,1);
					H(3,2) += (double)hipblasSdot(unSizeOfSec,fCenScnZ+i*unSizeOfSec,1,fCenModY+i*unSizeOfSec,1);
					H(3,3) += (double)hipblasSdot(unSizeOfSec,fCenScnZ+i*unSizeOfSec,1,fCenModZ+i*unSizeOfSec,1);
				}
				if(unStub){
					H(1,1) += (double)hipblasSdot(unStub,fCenScnX+unSections*unSizeOfSec,1,fCenModX+unSections*unSizeOfSec,1);
					H(1,2) += (double)hipblasSdot(unStub,fCenScnX+unSections*unSizeOfSec,1,fCenModY+unSections*unSizeOfSec,1);
					H(1,3) += (double)hipblasSdot(unStub,fCenScnX+unSections*unSizeOfSec,1,fCenModZ+unSections*unSizeOfSec,1);
					H(2,1) += (double)hipblasSdot(unStub,fCenScnY+unSections*unSizeOfSec,1,fCenModX+unSections*unSizeOfSec,1);
					H(2,2) += (double)hipblasSdot(unStub,fCenScnY+unSections*unSizeOfSec,1,fCenModY+unSections*unSizeOfSec,1);
					H(2,3) += (double)hipblasSdot(unStub,fCenScnY+unSections*unSizeOfSec,1,fCenModZ+unSections*unSizeOfSec,1);
					H(3,1) += (double)hipblasSdot(unStub,fCenScnZ+unSections*unSizeOfSec,1,fCenModX+unSections*unSizeOfSec,1);
					H(3,2) += (double)hipblasSdot(unStub,fCenScnZ+unSections*unSizeOfSec,1,fCenModY+unSections*unSizeOfSec,1);
					H(3,3) += (double)hipblasSdot(unStub,fCenScnZ+unSections*unSizeOfSec,1,fCenModZ+unSections*unSizeOfSec,1);
				}
			}
			getCublasErr();
                        return H;

}

void CIcpGpuCuda::printMatrix(Matrix* mat){
    for(int i = 1 ; i < 5 ; ++i)
        cout<<(*mat)(i,1)<< " "<<(*mat)(i,2)<< " "<<(*mat)(i,3)<<
            " "<<(*mat)(i,4)<< endl;
}
