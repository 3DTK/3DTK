#include "hip/hip_runtime.h"
/** @file
 *  @brief GPU kernel operation
 *  @author Deyuan Qii, University of Applied Sciences Bonn-Rhein-Sieg, Sankt Augustin, Germany.
 *                      Fraunhofer IAIS, Sankt Augustin, Germany.
 */


#define nns_kernel(QSelector)	__global__ void nns_priority##QSelector(\
		float* fDevScnX,\
		float* fDevScnY,\
		float* fDevScnZ,\
		float* fDist,\
		float* fDevSplit,\
		unsigned* unDevIdx,\
		unsigned* unDevAxis,\
		bool* bDevIsLeaf,\
		float* fDevLoBound,\
		float* fDevHiBound,\
		unsigned* unMask,\
		float* fDevMdlPairX,\
		float* fDevMdlPairY,\
		float* fDevMdlPairZ,\
		float* fDevScnPairX,\
		float* fDevScnPairY,\
		float* fDevScnPairZ,\
		float fSearchRadius,\
		unsigned unSize,\
		unsigned unWidth){\
\
/*/////////////*/\
/* preparation */\
/*/////////////*/\
	const unsigned unSizeData = unSize;\
	const int nWidth = (int)unWidth;\
	const unsigned tid = blockIdx.x*blockDim.x + threadIdx.x;\
	const unsigned unQSize = (unsigned)QSIZE##QSelector;\
	const float fRadius = fSearchRadius*fSearchRadius;\
\
/*////////*/\
/* kernel */\
/*////////*/\
	if(tid<unSizeData){\
\
	float fQ[3] = {fDevScnX[tid], fDevScnY[tid], fDevScnZ[tid]}; 		/* extract query point to register*/\
	float fM[3] = {0.0f, 0.0f, 0.0f};\
\
	unsigned Q_Idx[unQSize];\
	unsigned* unQ_Idx = &Q_Idx[0];\
	float Q_Dist[unQSize];\
	float* fQ_Dist = &Q_Dist[0];\
	Q q(unQ_Idx, fQ_Dist, unQSize);\
	float nn_dist = HUGE; 												/* initial distance*/\
	unsigned node = 0;\
	unsigned* u = &node;\
	float fDistance = 0.0;\
	float* rd = &fDistance; 											/* distance to rectangle*/\
	q.Insert(1, 0.0); 													/* start with root of tree*/\
\
	unsigned unNode = 0;\
	unsigned unNodeIdx = 0;\
	float fRd = 0.0f;\
	unsigned cd = 0;\
	float old_off, new_rd;\
	float new_off = 0.0f;\
	unsigned unPointIdx = 0;\
	int nPointIdxX = 0;\
	int nPointIdxY = 0;\
	float4 f4Mdl;\
	float fM_temp[3];\
	float fDist_temp = 0.0f;\
\
	while (q.notEmpty()) { 												/* repeat until queue is empty*/\
		q.Extr_Min(u, rd); 												/* closest node to query point*/\
		unNode = *u;													/* extract data to register*/\
		unNodeIdx = unNode-1;\
		fRd = *rd;\
		if (fRd >= nn_dist)						 						/* further from nearest so far*/\
			break;\
		while (!bDevIsLeaf[unNodeIdx]) { 								/* descend until leaf found*/\
			cd = unDevAxis[unNodeIdx]; 									/* cutting dimension*/\
			new_off = fQ[cd] - fDevSplit[unNodeIdx]; 					/* offset to further child*/\
			if (new_off < 0) { 											/* q is below cutting plane*/\
				old_off = fQ[cd] - fDevLoBound[unNodeIdx]; 				/* compute offset*/\
				if (old_off > 0) 										/* overlaps interval*/\
					old_off = 0;\
				new_rd = fRd - old_off*old_off 							/* distance to further child*/\
						+ new_off*new_off;\
				q.Insert(unNode*2+1, new_rd); 							/* enqueue hi_child for later*/\
				unNode *= 2; 											/* visit lo_child next*/\
				unNodeIdx = unNode-1;\
			}\
			else { 														/* q is above cutting plane*/\
				old_off = fQ[cd] - fDevHiBound[unNodeIdx];\
				if(old_off < 0)\
					old_off = 0;\
				new_rd = fRd - old_off*old_off 							/* distance to further child*/\
						+ new_off*new_off;\
				q.Insert(unNode*2, new_rd);\
\
				unNode = 2*unNode + 1;\
				unNodeIdx = unNode-1;\
			}\
		}\
		unPointIdx = unDevIdx[unNodeIdx];\
		nPointIdxX=(int)unPointIdx%nWidth;								/*access via texture*/\
		nPointIdxY=(int)unPointIdx/nWidth;\
		f4Mdl=tex2D(refTex,(float)nPointIdxX,(float)nPointIdxY);\
		fM_temp[0]=f4Mdl.x;\
		fM_temp[1]=f4Mdl.y;\
		fM_temp[2]=f4Mdl.z;\
		fDist_temp = DIST(fM_temp,fQ);\
		if(fDist_temp<nn_dist){\
			nn_dist = fDist_temp;\
			fM[0]=fM_temp[0];\
			fM[1]=fM_temp[1];\
			fM[2]=fM_temp[2];\
		}\
	}\
\
\
	if(nn_dist>fRadius){												/*if non-pair*/\
		unMask[tid]=0;\
		fDevMdlPairX[tid]=0.0f;	fDevMdlPairY[tid]=0.0f;	fDevMdlPairZ[tid]=0.0f;\
		fDevScnPairX[tid]=0.0f;	fDevScnPairY[tid]=0.0f;	fDevScnPairZ[tid]=0.0f;\
	}\
	else{\
		unMask[tid]=1;\
		fDist[tid] = nn_dist;			/*return nearest distance for deviation calculation*/\
		fDevMdlPairX[tid]=fM[0];	fDevMdlPairY[tid]=fM[1];	fDevMdlPairZ[tid]=fM[2];\
		fDevScnPairX[tid]=fQ[0];	fDevScnPairY[tid]=fQ[1];	fDevScnPairZ[tid]=fQ[2];\
	}\
	}\
}

nns_kernel(1)
nns_kernel(2)
nns_kernel(3)
nns_kernel(4)
nns_kernel(5)
nns_kernel(6)

__global__ void centralize(unsigned* unMask,
		float* fDevMdlPairX,
		float* fDevMdlPairY,
		float* fDevMdlPairZ,
		float* fDevScnPairX,
		float* fDevScnPairY,
		float* fDevScnPairZ,
		float fcm0,
		float fcm1,
		float fcm2,
		float fcs0,
		float fcs1,
		float fcs2,
		float* fCenteredModX,
		float* fCenteredModY,
		float* fCenteredModZ,
		float* fCenteredScnX,
		float* fCenteredScnY,
		float* fCenteredScnZ){
	const unsigned tid = blockIdx.x*blockDim.x + threadIdx.x;

	if(unMask[tid]){
		fCenteredModX[tid] = fDevMdlPairX[tid] - fcm0;
		fCenteredModY[tid] = fDevMdlPairY[tid] - fcm1; 
		fCenteredModZ[tid] = fDevMdlPairZ[tid] - fcm2; 
		fCenteredScnX[tid] = fDevScnPairX[tid] - fcs0;
		fCenteredScnY[tid] = fDevScnPairY[tid] - fcs1; 
		fCenteredScnZ[tid] = fDevScnPairZ[tid] - fcs2; 
	}else{
		fCenteredModX[tid] = 0.0f;
		fCenteredModY[tid] = 0.0f; 
		fCenteredModZ[tid] = 0.0f; 
		fCenteredScnX[tid] = 0.0f;
		fCenteredScnY[tid] = 0.0f; 
		fCenteredScnZ[tid] = 0.0f; 
	}
}


__global__ void transformation(float* fDevScnX,
		float* fDevScnY,
		float* fDevScnZ,
		float m00, float m01, float m02, float m03,
		float m10, float m11, float m12, float m13,
		float m20, float m21, float m22, float m23){
	const unsigned tid = blockIdx.x*blockDim.x + threadIdx.x;
	double adTmp[3];
	adTmp[0] = fDevScnX[tid] * m00 +
			   fDevScnY[tid] * m01 +
			   fDevScnZ[tid] * m02 + m03;
	adTmp[1] = fDevScnX[tid] * m10 +
			   fDevScnY[tid] * m11 +
			   fDevScnZ[tid] * m12 + m13;
	adTmp[2] = fDevScnX[tid] * m20 +
			   fDevScnY[tid] * m21 +
			   fDevScnZ[tid] * m22 + m23;
	fDevScnX[tid] = adTmp[0];
	fDevScnY[tid] = adTmp[1];
	fDevScnZ[tid] = adTmp[2];		
}
